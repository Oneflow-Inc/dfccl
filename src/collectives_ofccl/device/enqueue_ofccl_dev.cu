#include "hip/hip_runtime.h"
#include "enqueue_ofccl_dev.h"

// TODO: nccl最新的代码里，这部分的设计和实现都变了。
//
// Copy src to dst and fill extra size with zeroes
// 这个是保证在一次调用复制完最多512B，并且以16B为单位。
// 这个不要求src dst同一类型
template<typename Tdst, typename Tsrc>
static __device__ void copyToShmemOneShot(Tdst *dst, Tsrc const *src, int tid, int nthreads) { // nccl的这个的函数签名里有个nthreads参数，但是并没有用，应该是为了和下边那个作区分，现在我们可以区分开了，反而带上nthreads是区分不开的。
  static_assert(sizeof(Tdst)%(2*sizeof(uint64_t )) == 0 && sizeof(Tsrc)%(2*sizeof(uint64_t)) == 0,
      "copyToShmemOneShot needs sizes which are multiple of 16B");
  static_assert(sizeof(Tdst) >= sizeof(Tsrc), "Tdst size is too small");
  static_assert(sizeof(Tdst) <= WARP_SIZE*2*sizeof(uint64_t), "copyToShmemOneShot limited to 512B to make sure it can always be done in one cycle");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  uint64_t *shmemPtr = shmemCvtPtr_ofccl(d); // 由于这个地方，这个函数只能用于dst是shmem的情况了。
  int offset = 2*tid;
  uint64_t v0, v1;
  if (offset >= sizeof(Tsrc)/sizeof(uint64_t)) {
    v0 = v1 = 0ULL;
  } else {
    v0 = s[offset] ; v1 = s[offset+1];
  }
  if (offset < sizeof(Tdst)/sizeof(uint64_t)) storeShmem128_ofccl(shmemPtr+offset, v0, v1);
}

// 这个可以直接用到任意一轮搞不完的数据结构的复制吧。
// 这个要求src dst同一类型。
// turn的作用：
template<typename T>
static __device__ int copyToShmemLoop(T *dst, T const *src, int tid, int nthreads, int turn=0) {
  static_assert(sizeof(uint64_t) <= alignof(T), "Uhoh");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  int t = tid - turn;
  if (t < 0) t += nthreads;
  int n = sizeof(T)/sizeof(uint64_t); // n 代表要复制的数据结构包含了几个8Byte

  int delta = (n + WARP_SIZE-1) & -WARP_SIZE; // round up to warp lane 0; 要把n和WARP_SIZE处理对齐了。
  //  32 = 0000 0000 0010 0000
  // -32 = 1111 1111 1110 0000，低位不变，高位都置1。大一的东西忘却了。。
  // 所以delta相当于n相对于32的“向上取整”，即向上取到32的整数倍。

  if (delta < nthreads) { // 总的要传的 8Byte 的个数小于blockDim.x（我们的case里是thrdLimit）
    turn += delta;
    if (turn >= nthreads) turn -= nthreads; // 在第一次调用里这个不会成立，应该是为了后续的调用使用
  }
  else
    turn = 0; // 如果总的要传的 8Byte 的个数超过了blockDim.x，那就不用管turn了。所以turn就是为了雨露均沾，让所有线程都干活

  n -= t; // 对每个线程来说，砍掉比tid小的几项，不用自己管。
  d += t; // 对每个线程来说，自己从tid的偏移量开始管。
  s += t;
  #pragma unroll // 指示要循环展开。
  for (int i=0; i < divUp(sizeof(T), WARP_SIZE*sizeof(uint64_t)); i++) {
    if (n > 0) {
      *d = *s;
      d += nthreads;
      s += nthreads;
      n -= nthreads; // “一轮”完成 nthreads个8 Byte的复制。
    }
  }
  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> hello", sharedCollCtx.rank, blockIdx.x, tid);
  return turn;
}

// 这个的目的应该是在“切片并行复制”之后，恢复标量的语义
// 但是没用，而且在buffer里的数据是0.5，或者其他数字时，导致卡住。log发现buffer里的数字是0.25，可以正常运行，并且没有进入这里。所以直接注释了吧。
// TODO: 但是这里卡住，总还是怪怪的。有空看看吧。
// static __device__ void ofcclRedopPtrDeref(struct ncclWorkElem* we) {
//   if (we->header.type != ncclWorkTypeUnused && we->redOpArgIsPtr) {
//     /* redOpArg is a pointer to the scalar value, so we'll dereference it
//      * here so that redOpArg holds the bits of the scalar going forward.
//      * The tricky thing is we don't know its type T since that's encoded in
//      * the funcIndex. Because it would be difficult to get sizeof(T) from
//      * funcIndex, we'll cheat and just dereference the largest possible size
//      * given the alignment of the pointer. We might be reading in more bytes
//      * than we need but that's harmless.
//      */
//     if (we->redOpArg%2 != 0)
//       we->redOpArg = *reinterpret_cast<uint8_t*>(we->redOpArg);
//     else if (we->redOpArg%4 != 0)
//       we->redOpArg = *reinterpret_cast<uint16_t*>(we->redOpArg);
//     else if (we->redOpArg%8 != 0)
//       we->redOpArg = *reinterpret_cast<uint32_t*>(we->redOpArg);
//     else
//       we->redOpArg = *reinterpret_cast<uint64_t*>(we->redOpArg);
//     // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, we->redOpArgIsPtr = %d, we->redOpArg = %llu", sharedCollCtx.rank, blockIdx.x, threadIdx.x, we->redOpArgIsPtr, we->redOpArg);
//   }
// }

// share mem用超了。
// TODO: 可以不同的algo、proto使用不同的数据类型，不过可以看看是不是有意义
__shared__ CollCtx sharedCollCtx; // 不能static，primitives要用

__shared__ BlkStatus blkStatus; // 取消static，放到prim里边打印log。
// TODO: 下边这几个可以尝试用constant，先不急
static __shared__ int sharedBlkCount4Coll[MAX_LENGTH];
// static __shared__ int sharedThrdCount4Coll[MAX_LENGTH];

static __device__ int sqRead(SQ *sq, SQE *target, int thrdCudaDev) {

  unsigned long long int currSqFrontier = blkStatus.sqReadFrontier;

  // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, enter, sqReadFrontier = %llu, sq->head=%llu, sq->tail=%llu", thrdCudaDev, blockIdx.x, threadIdx.x, DevRingBufferLogicFrontier(sq, currSqFrontier), DevLogicSqHead(sq), DevLogicSqTail(sq)); // sharedCollCtx.rank是在loadCtx之后才有效的，在此之前想打印sqRead的情况，需要使用thrdCudaDev，不然会搞出乌龙。

  if (DevSqEmpty(sq, currSqFrontier)) {
    return -1;
  }
  // 先读过来，然后再判断，最后更新状态：sqe->counter; 以及在恰当的时候commit read
  *target = *DevRingBufferGetFrontier(sq, currSqFrontier);
  if (target->quit) {
    // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Get quit", thrdCudaDev, bid, threadIdx.x);
    return 0;
  }

  int oldCounter = atomicAdd(&(DevRingBufferGetFrontier(sq, currSqFrontier)->counter), 1); // 将自己读了的sqe的counter加1，代表有人读过了，有一个block不需要再读这个sqe了，后来再有人读这个的时候加完了去判断。

  blkStatus.sqReadFrontier++; // 这次读到了，那对于当前这个block来说，下一个可读的位置前进一个。

  // OFCCL_LOG_RANK_X(OFCCL, 0, "Rank<%d> Blk<%d> Thrd<%d>, update counter = %d for coll_id = %d, @ %llu", thrdCudaDev, blockIdx.x, threadIdx.x, oldCounter + 1, DevRingBufferGetFrontier(sq, currSqFrontier)->collId, DevRingBufferLogicFrontier(sq, currSqFrontier));

  __threadfence(); // 保证device上的各个block不要乱序看到。

  unsigned long long int sqHead;
  if (oldCounter + 1 == gridDim.x) {
    do {
      sqHead = atomicCAS(&sq->head, currSqFrontier, currSqFrontier + 1);
    } while (sqHead != currSqFrontier);

    // OFCCL_LOG_RANK_X(OFCCL, 0, "Rank<%d> Blk<%d> Thrd<%d>, update sq->head, blkStatus.sqReadFrontier = %llu, sq->head = %llu", thrdCudaDev, blockIdx.x, threadIdx.x, DevRingBufferLogicFrontier(sq, blkStatus.sqReadFrontier), DevLogicSqHead(sq));
  }

  return 0;
}

static __device__ int cqWrite(CQ *cq, CQE *cqe, int thrdCudaDev, unsigned long long int *cqeWriteCnt) {
  if (DevCqFull(cq)) {
    // not an error; caller keeps trying.
    return -1;
  }

  unsigned long long int myCqFrontier = atomicAdd(&(cq->frontier), 1); // 占坑，我就往这里写了，用的是old值，新的cq->tail预期是atomicAdd之后的cq->frontier，也就是myCqFrontier + 1。
  // 两个线程同时调用atomicAdd，是严格保证各自返回的。

  // *(blkStatus.collCounters + 5 + cqe->collId * COLL_COUNTER_INNER_SIZE + blockIdx.x * MAX_LENGTH * COLL_COUNTER_INNER_SIZE) = DevRingBufferLogicFrontier(cq, myCqFrontier);
  // *(blkStatus.collCounters + 6 + cqe->collId * COLL_COUNTER_INNER_SIZE + blockIdx.x * MAX_LENGTH * COLL_COUNTER_INNER_SIZE) = cq->tail;

  __threadfence();

  DevRingBufferGetFrontier(cq, myCqFrontier)->collId = cqe->collId; // 那这里也应该各自写进去了。

  __threadfence_system();

  // atomicCAS返回地址上的old值，是否修改体现不在返回值上。
  unsigned long long int cqTail;
  do {
    cqTail = atomicCAS(&cq->tail, myCqFrontier, myCqFrontier + 1);
  } while(cqTail != myCqFrontier); // while这里是观察CAS里的条件是否被满足，如果观察到这个条件满足了，那也就可以确定Swap的操作也就完成了。

  // *(blkStatus.collCounters + 1 + cqe->collId * COLL_COUNTER_INNER_SIZE + blockIdx.x * MAX_LENGTH * COLL_COUNTER_INNER_SIZE) += 1;
  #ifdef CQE_DEBUG_RANK_X
    OFCCL_LOG_RANK_X(OFCCL_CQE, CQE_DEBUG_RANK_X, "Rank<%d> Blk<%d> Thrd<%d>, put %lluth CQE for coll_id = %d @ %llu and update cq->tail", thrdCudaDev, blockIdx.x, threadIdx.x, ++(*cqeWriteCnt), cqe->collId, DevRingBufferLogicFrontier(cq, myCqFrontier));
  #endif
  #ifdef CQE_DEBUG_ALL_RANK
    OFCCL_LOG(OFCCL_CQE, "Rank<%d> Blk<%d> Thrd<%d>, put %lluth CQE for coll_id = %d @ %llu and update cq->tail", thrdCudaDev, blockIdx.x, threadIdx.x, ++(*cqeWriteCnt), cqe->collId, DevRingBufferLogicFrontier(cq, myCqFrontier));
  #endif
  return 0;
}

// TODO: 为了性能，考虑恢复成多线程一起复制的写法。
static __device__ void copyNcclWorkElem (struct ncclWorkElem &dstElem, const struct ncclWorkElem &srcElem) {
  dstElem.header.funcIndex = srcElem.header.funcIndex;
  dstElem.header.type = srcElem.header.type;
  dstElem.header.nWarps = srcElem.header.nWarps;
  dstElem.header.isLast = srcElem.header.isLast;

  dstElem.regUsed = srcElem.regUsed;
  dstElem.direct = srcElem.direct;
  dstElem.redOpArgIsPtr = srcElem.redOpArgIsPtr;
  dstElem.sendbuff = srcElem.sendbuff;
  dstElem.recvbuff = srcElem.recvbuff;
  dstElem.count = srcElem.count;
  dstElem.lastChunkSize = srcElem.lastChunkSize;
  dstElem.root = srcElem.root;
  dstElem.bid = srcElem.bid;
  dstElem.nChannels = srcElem.nChannels;
  dstElem.redOpArg = srcElem.redOpArg;
}

static __device__ int initContexts(int thrdCudaDev, int collCount, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx, int turn) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  // int nthreads = blockDim.x;
  // 构建任务列表
  // TODO: 并行提高复制效率。
  if (tid == 0) {
    for (int i = 0; i < collCount; i++) {
      int collId = globalCollIds[i];
      // 以下这两个变量会限制很多行为。
      int blkLimit = sharedBlkCount4Coll[collId] = globalBlkCount4Coll[collId];
      // sharedThrdCount4Coll[collId] = globalThrdCount4Coll[collId];

      // 下边这部分逻辑在在blkStatus.hasVolunteerQuitted == 1的情况下不执行，曾经退出过，恢复的时候就不要重新初始化了。
      if (blkStatus.hasVolunteerQuitted == 0) {
        // 每个block一份globalShmem
        CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;

        // ***** 移植ncclKernel的逻辑 *****
        if (bid < blkLimit) {
          // ncclDevComm *comm = globalDevComm7WorkElems[collId].comm;
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->comm), comm, tid, nthreads, turn);
          // // 一个奇技淫巧：get address of channel without incurring indirect load from ncclDevComm::channels
          // // 这里通过bid选择了合适的channel，很多集合通信真正执行时用到的硬件信息就存在channel里边。
          // ncclChannel *channel = &((ncclDevCommAndChannels*)comm)->channels[bid];
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->channel), channel, tid, nthreads, turn); // 尝试使用oneshot，会报错warp misaligned，所以看来必须用loop。

          // // nccl中限制只在bid=0里进行这样的拷贝，对于ofccl而言，ofcclShmem就是任务列表，所以对于所有的线程，我们都把同样的work存进去；
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->work.elems[0]), &(globalDevComm7WorkElems[collId].first), tid, nthreads, turn); // nccl 2.12里边这地方用copyToShmemOneShot进行拷贝，但是oneShot的实现使用了与shared mem相关的内联汇编，所以这里也使用loop进行拷贝。
          // // nccl中接下来要处理channel.workFifoDev，然而对于目前的ofccl，只处理first就好，channel.workFifoDev不会有其他任务了。
          // __syncthreads(); // 等待全部线程加载完成

          // if (globalCollCtx4Blk7Coll->work.header.type == ncclWorkTypeColl) {
          //   // #define NCCL_MAX_WORK_ELEMENTS (NCCL_WORK_SIZE / sizeof(struct ncclWorkElem))=512/64=8
          //   // 原来这个写法，应该是想修改we->redOpArg，不过修改we->redOpArg一个线程就够了，所以让理论上最多的线程来工作，咱们保留就好。
          //   if (tid < NCCL_MAX_WORK_ELEMENTS) ofcclRedopPtrDeref(&(globalCollCtx4Blk7Coll->work.elems[tid]));
          // } // 目前不用考虑其他ncclWorkType
          // __syncthreads();

          /* ****** 手动加载用得到的shmemData ****** */
          ncclDevComm *comm = globalDevComm7WorkElems[collId].comm;
          ncclChannel *channel = &((ncclDevCommAndChannels*)comm)->channels[bid];

          globalCollCtx4Blk7Coll->ringPrev = channel->ring.prev;
          globalCollCtx4Blk7Coll->ringNext = channel->ring.next;
          globalCollCtx4Blk7Coll->ringIndex = channel->ring.index;
          globalCollCtx4Blk7Coll->devPeers = channel->devPeers; // 直接赋值指针

          globalCollCtx4Blk7Coll->rank = comm->rank;
          globalCollCtx4Blk7Coll->nRanks = comm->nRanks;
          globalCollCtx4Blk7Coll->abortFlag = comm->abortFlag;

          for (int i = 0; i < NCCL_NUM_PROTOCOLS; i++) {
            globalCollCtx4Blk7Coll->buffSizes[i] = comm->buffSizes[i];
          }

          copyNcclWorkElem(globalCollCtx4Blk7Coll->workElem, globalDevComm7WorkElems[collId].first);

          /* ****** 上下文 ****** */

          #if defined(CQE_DEBUG_RANK_X) || defined(CQE_DEBUG_ALL_RANK)
            globalCollCtx4Blk7Coll->sqeReadCnt = 0;
            globalCollCtx4Blk7Coll->cqePrepareCnt = 0;
            globalCollCtx4Blk7Coll->cqeWriteCnt = 0;
          #endif

          // bugfix: 下边原来都是设置的globalBlk2CollId2CollCtx->XXXX，相当于都设置了第0个block的第0个coll。。。。。。。
          globalCollCtx4Blk7Coll->loadAgain = 0;
          globalCollCtx4Blk7Coll->slice4SimpleGenericOp = 0;
          globalCollCtx4Blk7Coll->offset4SimpleGenericOp = 0;

          globalCollCtx4Blk7Coll->currentStep4RingAllReduce = 0;
          globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = 0;
        }
      }
    }
  }
  ofcclBarrier(1);
  return turn;
}

static __device__ void logTaskQ(int caller, int thrdCudaDev, int rank=-1) {
  if (rank == -1) {
    rank = thrdCudaDev;
  }
  OFCCL_LOG_RANK_X(OFCCL_CQE, rank, "Rank<%d> Blk<%d> Thrd<%d>, caller = %d, numActiveColls=%d, TaskQ: [%d-%d-%d-%d-%d-%d-%d-%d-%d-%d]", thrdCudaDev, blockIdx.x, threadIdx.x, caller, blkStatus.numActiveColls, blkStatus.activeCollIds[0], blkStatus.activeCollIds[1], blkStatus.activeCollIds[2], blkStatus.activeCollIds[3], blkStatus.activeCollIds[4], blkStatus.activeCollIds[5], blkStatus.activeCollIds[6], blkStatus.activeCollIds[7], blkStatus.activeCollIds[8], blkStatus.activeCollIds[9]);
}

// 这个是有必要的，在没看到全部都要退出这个信息之前，还是可以撤回自己要退出的标志。
static __device__ void cancelQuit(int *globalVolunteerQuitCounter) {
  if (blkStatus.iWantToQuit) { // 如果我之前投票要退出，现在立刻撤回。
    blkStatus.iWantToQuit = false;
    blkStatus.seenAllBlockWantToQuitCounter = 0; // 保证我自己肯定不退了
    atomicAdd(globalVolunteerQuitCounter, -1);
  }
}

// 为了初步实现按需启停，增加一个“空read计数，读不到新的，增加计数”
static __device__ void checkSQ7TidyTaskQ(int thrdCudaDev, SQ *sq, CollCtx *globalBlk2CollId2CollCtx, int *failCnt, int *finallyQuit, int *globalVolunteerQuitCounter, int *unprogressedCnt) {
  int bid = blockIdx.x;

  SQE target;

  // 能读到，假如是正常SQE，把信息在任务列表里记录一下；假如是quit，那也记录一下
  // 读不到新东西那就算了

  if (sqRead(sq, &target, thrdCudaDev) == -1) {
    *failCnt += 1;
    *unprogressedCnt += 1;
    if (blkStatus.numActiveColls > 0) {
      *failCnt = 0; // TODO: 更改failCnt的更新逻辑，觉得自己死锁了，虽然任务列表不空，但是半天动不了，也可以退。
      // 这种情况不重置unprogressedCnt，识别本地任务列表不空，但是无法推进的情况。
      cancelQuit(globalVolunteerQuitCounter);
      
      // 没读到新的，应该不用处理taskQ了，因为每次遍历一次taskQ，都会处理。 
    }
    return;
  } else {
    // TODO: 更改failCnt的更新逻辑，觉得自己死锁了，虽然任务列表不空，但是半天动不了，也可以退。
    *failCnt = 0;
    *unprogressedCnt = 0;
    cancelQuit(globalVolunteerQuitCounter);
    if (target.quit) {
      blkStatus.quit = 1; // TODO: 从鲁棒性的角度来说，这里应该有机制保证看到这个quit sqe的时候，taskQ里的所有sqe也应该都处理完，才能退出。（不过目前可以先不管，可以由用户程序间接保证）；一个简单的保证方法是，加一个check。
      // if (bid == 0) {
        *finallyQuit = 1; // TODO: 为了最后每个block都保证打印统计信息，挺不优雅的
      // }
      return;
    }

    // 正常读到了SQE的话，需要往global的globalBlk2CollId2CollCtx表项里边写入，更新blkStatus.numActiveColls
    int newActiveCollId = target.collId;
    int blkLimit = sharedBlkCount4Coll[newActiveCollId]; // 需要参与新读到的coll的block才会进行后续操作。
    if (bid < blkLimit) {
      CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + newActiveCollId;
      // if (blkStatus.collStatus[newActiveCollId] != 0) { // 应该没有重入的风险。重入指一个正在执行的集合通信又被提起请求。
      //   OFCCL_LOG(OFCCL_FATAL, "Rank<%d> Blk<%d> Thrd<%d> globalCollCtx4Blk7Coll->executing should be 0! sq->head = %llu, sq->tail = %llu, blkStatus.sqReadFrontier = %llu", thrdCudaDev, bid, threadIdx.x, DevLogicSqHead(sq), DevLogicSqTail(sq), DevRingBufferLogicFrontier(sq, blkStatus.sqReadFrontier));
      // }

      blkStatus.collStatus[newActiveCollId] = 1;
      OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, set blkStatus.collStatus[%d]=1;", thrdCudaDev, blockIdx.x, threadIdx.x, newActiveCollId);
      
      #ifdef CQE_DEBUG_RANK_X
        OFCCL_LOG_RANK_X(OFCCL_CQE, CQE_DEBUG_RANK_X, "Rank<%d> Blk<%d> Thrd<%d>, read %lluth SQE for coll_id = %d, sq->head = %llu, sq->tail = %llu, blkStatus.sqReadFrontier = %llu", thrdCudaDev, blockIdx.x, threadIdx.x, ++(globalCollCtx4Blk7Coll->sqeReadCnt), newActiveCollId, DevLogicSqHead(sq), DevLogicSqTail(sq), DevRingBufferLogicFrontier(sq, blkStatus.sqReadFrontier));
      #endif
      #ifdef CQE_DEBUG_ALL_RANK
        OFCCL_LOG(OFCCL_CQE, "Rank<%d> Blk<%d> Thrd<%d>, read %lluth SQE for coll_id = %d, sq->head = %llu, sq->tail = %llu, blkStatus.sqReadFrontier = %llu", thrdCudaDev, blockIdx.x, threadIdx.x, ++(globalCollCtx4Blk7Coll->sqeReadCnt), newActiveCollId, DevLogicSqHead(sq), DevLogicSqTail(sq), DevRingBufferLogicFrontier(sq, blkStatus.sqReadFrontier));
      #endif
      
      globalCollCtx4Blk7Coll->workElem.sendbuff = target.sendbuff;
      globalCollCtx4Blk7Coll->workElem.recvbuff = target.recvbuff;

      // maintain the taskQ here.
      // 新加入的集合通信放在末位，最后执行。如果新加入的集合通信存在于当前的blkStatus.activeCollIds里边，也不必强行放到末位。
      int new_numActiveColls = 0;
      bool newActiveCollId_in_taskQ = false;
      // TODO: 考虑循环展开的优化。
      for (int i = 0; i < blkStatus.numActiveColls; ++i) {
        int collIdInTaskQ = blkStatus.activeCollIds[i];
        if (collIdInTaskQ == newActiveCollId) {
          newActiveCollId_in_taskQ = true;
        }
        if (blkStatus.collStatus[collIdInTaskQ] != 0) { // 1 正在执行和 -1 switch都算在执行中，要保留在任务列表中，应该不会有2
          // 在同一个数组上就地操作。new_numActiveColls一定是<=i的，所以不会有问题。
          blkStatus.activeCollIds[new_numActiveColls++] = collIdInTaskQ;
        }
      }
      if (!newActiveCollId_in_taskQ) {
        blkStatus.activeCollIds[new_numActiveColls++] = newActiveCollId;
      }

      blkStatus.numActiveColls = new_numActiveColls;
      #ifdef CQE_DEBUG_ALL_RANK
        logTaskQ(0, thrdCudaDev, -1);
      #elif defined(CQE_DEBUG_RANK_X)
        logTaskQ(0, thrdCudaDev, CQE_DEBUG_RANK_X);
      #endif
    }
  }
}

static __device__ int loadCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int collId, int turn, int64_t BASE_CTX_SWITCH_THRESHOLD) {
  int tid = threadIdx.x;
  // int nthreads = blockDim.x;

  // turn = copyToShmemLoop(&sharedCollCtx.comm, &(globalCollCtx4Blk7Coll->comm), tid, nthreads, turn);
  // turn = copyToShmemLoop(&sharedCollCtx.channel, &(globalCollCtx4Blk7Coll->channel), tid, nthreads, turn);
  // // copyToShmemOneShot(&sharedCollCtx.work, &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads); // TODO: 用了这个会报错misaligned，就先loop吧
  // turn = copyToShmemLoop(&(sharedCollCtx.work.elems[0]), &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads, turn);
  // sharedCollCtx.work.elems[0].header.nWarps = globalCollCtx4Blk7Coll->work.elems[0].header.nWarps;
  // // turn = copyToShmemLoop(&sharedCollCtx.work, &(globalCollCtx4Blk7Coll->work), tid, nthreads, turn);
  // __syncthreads(); // 全部线程都执行，可以使用这个同步。

  if (tid == 0) {
    sharedCollCtx.ringPrev = globalCollCtx4Blk7Coll->ringPrev;
    sharedCollCtx.ringNext = globalCollCtx4Blk7Coll->ringNext;
    sharedCollCtx.ringIndex = globalCollCtx4Blk7Coll->ringIndex;
    sharedCollCtx.devPeers = globalCollCtx4Blk7Coll->devPeers;

    sharedCollCtx.rank = globalCollCtx4Blk7Coll->rank;
    sharedCollCtx.nRanks = globalCollCtx4Blk7Coll->nRanks;
    sharedCollCtx.abortFlag = globalCollCtx4Blk7Coll->abortFlag;

    for (int i = 0; i < NCCL_NUM_PROTOCOLS; i++) {
      sharedCollCtx.buffSizes[i] = globalCollCtx4Blk7Coll->buffSizes[i];
    }

    copyNcclWorkElem(sharedCollCtx.workElem, globalCollCtx4Blk7Coll->workElem);

    // // for debug
    // {
    //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
    //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
    //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
    //   uint64_t head = recvConn->step;
    //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
    //   uint64_t tail = sendConn->step;
    //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d load head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, threadIdx.x, collId, head, tail);
    // }

    // 加载algo、proto、func相关的运行上下文。
    // TODO: 目前只有simple ring allreduce，之后考虑通用性和扩展性。
    blkStatus.currLoadedCollId = collId;
    
    sharedCollCtx.loadAgain = globalCollCtx4Blk7Coll->loadAgain;
    sharedCollCtx.slice4SimpleGenericOp = globalCollCtx4Blk7Coll->slice4SimpleGenericOp;
    sharedCollCtx.offset4SimpleGenericOp = globalCollCtx4Blk7Coll->offset4SimpleGenericOp;

    // sharedCollCtx.totalSteps4RingAllReduce = 2 * sharedCollCtx.nRanks - 1;
    sharedCollCtx.currentStep4RingAllReduce = globalCollCtx4Blk7Coll->currentStep4RingAllReduce;
    sharedCollCtx.gridOffset4RingAllReduce = globalCollCtx4Blk7Coll->gridOffset4RingAllReduce;
    sharedCollCtx.ctxSwitchThreshold = BASE_CTX_SWITCH_THRESHOLD;
    // __threadfence_block();
  }
  // *(blkStatus.barrierCnt + 0 + 6 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  ofcclBarrier(2);
  // *(blkStatus.barrierCnt + 1 + 6 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;

  return turn;
}

static __device__ void manipulateCQ7ResetDoneColl(int thrdCudaDev, int doneCollId, CQ *cq, CQE *globalCqes, CollCtx *globalCollCtx4Blk7Coll, CollCtx *globalBlk2CollId2CollCtx) {
  // 协调所有blk，发现所有blk都完成，最后一个blk发送CQE
  int old_counter = atomicAdd(&(globalCqes[doneCollId].counter), 1);
  __threadfence(); // cqes在global memory里边，全部block关心。

  // *(blkStatus.collCounters + 0 + doneCollId * COLL_COUNTER_INNER_SIZE + blockIdx.x * MAX_LENGTH * COLL_COUNTER_INNER_SIZE) += 1;

  OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, prepare %lluth CQE for coll_id = %d", thrdCudaDev, blockIdx.x, threadIdx.x, ++(globalCollCtx4Blk7Coll->cqePrepareCnt), doneCollId);

  if (old_counter + 1 == sharedBlkCount4Coll[doneCollId]) {
    atomicExch(&globalCqes[doneCollId].counter, 0);

    #if defined(CQE_DEBUG_RANK_X) || defined(CQE_DEBUG_ALL_RANK)
      CollCtx *globalCollCtx4Blk_0_7Coll = globalBlk2CollId2CollCtx + 0 * MAX_LENGTH + doneCollId;
      unsigned long long int *cqeWriteCnt = &globalCollCtx4Blk_0_7Coll->cqeWriteCnt;
      while (cqWrite(cq, globalCqes + doneCollId, thrdCudaDev, cqeWriteCnt) == -1) {
      }
    #else
      while (cqWrite(cq, globalCqes + doneCollId, thrdCudaDev, nullptr) == -1) {
      }
    #endif
    // *(blkStatus.collCounters + 1 + doneCollId * COLL_COUNTER_INNER_SIZE + blockIdx.x * MAX_LENGTH * COLL_COUNTER_INNER_SIZE) += 1;
    __threadfence();
  }

  // 这里不再给blkStatus.numActiveColls减1，只给executing置0。
  blkStatus.currLoadedCollId = -1;

  blkStatus.collStatus[doneCollId] = 0;

  globalCollCtx4Blk7Coll->loadAgain = 0;

  // ResetDoneColl
  globalCollCtx4Blk7Coll->slice4SimpleGenericOp = 0;
  globalCollCtx4Blk7Coll->offset4SimpleGenericOp = 0;
  globalCollCtx4Blk7Coll->currentStep4RingAllReduce = 0;
  globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = 0;

  // for debug
  // {
  //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
  //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
  //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
  //   uint64_t head = recvConn->step;
  //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
  //   uint64_t tail = sendConn->step;
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d done head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, tid, doneCollId, head, tail);
  // }
}

static __device__ void saveExcutingCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int collId) {
  globalCollCtx4Blk7Coll->loadAgain = sharedCollCtx.loadAgain;
  globalCollCtx4Blk7Coll->slice4SimpleGenericOp = sharedCollCtx.slice4SimpleGenericOp;
  globalCollCtx4Blk7Coll->offset4SimpleGenericOp = sharedCollCtx.offset4SimpleGenericOp;

  globalCollCtx4Blk7Coll->currentStep4RingAllReduce = sharedCollCtx.currentStep4RingAllReduce;
  globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = sharedCollCtx.gridOffset4RingAllReduce;

  #ifdef SHOW_SWITCH_CNT
    blkStatus.totalCtxSwitchCnt++;
  #endif

  // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, blkStatus.totalCtxSwitchCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.totalCtxSwitchCnt, blkStatus.numActiveColls);

  // // for debug
  // {
  //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
  //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
  //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
  //   uint64_t head = recvConn->step;
  //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
  //   uint64_t tail = sendConn->step;
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d save head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, threadIdx.x, collId, head, tail);
  // }
}

static __device__ int maintainSharedCollCtx(int thrdCudaDev, CollCtx *globalBlk2CollId2CollCtx, int collId, int turn, int64_t BASE_CTX_SWITCH_THRESHOLD) {
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d, old blkStatus.currLoadedCollId=%d", thrdCudaDev, blockIdx.x, threadIdx.x, collId, blkStatus.currLoadedCollId);
  
  bool noLoadedColl = (blkStatus.currLoadedCollId == -1);
  bool sameLoadedColl = (collId == blkStatus.currLoadedCollId); // 在traverseTaskQ中遍历taskQ的情况下，其实只有在taskQ里只剩一个元素的时候，这个条件才可能成立
  // bool loadedCollDone = (blkStatus.collStatus[blkStatus.currLoadedCollId] == 2); // 这里不用关心这个。
  bool loadedCollSaveCtx7Quit = (blkStatus.collStatus[blkStatus.currLoadedCollId] == -1);

  bool needSave = !noLoadedColl && !sameLoadedColl && loadedCollSaveCtx7Quit;
  bool needLoad = noLoadedColl || !sameLoadedColl;

  ofcclBarrier(4);
  if (tid == 0) {
    // bugfix：不在这里重置，对于不需要save再load的-1的coll，就没机会重置了，进而在all_reduce.h里走不动。
    blkStatus.collStatus[collId] = 1; // 每次准备执行的时候，重置为正常执行状态。新的coll已经是1，不过不要浪费if了。
    sharedCollCtx.saveCtx7Quit = 0; // 重置。
    
    if (needSave) {
      // bugfix: save的时候，不应该save到即将load的coll的global collCtx副本里。
      CollCtx *globalCollCtx4Blk7OldColl = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + blkStatus.currLoadedCollId;

      OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> save ctx for coll_id = %d", thrdCudaDev, blockIdx.x, threadIdx.x, blkStatus.currLoadedCollId);
      saveExcutingCollCtx(thrdCudaDev, globalCollCtx4Blk7OldColl, blkStatus.currLoadedCollId);
    }
  }
  ofcclBarrier(11);

  if (needLoad) {
    CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;
    OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> load ctx for coll_id = %d", thrdCudaDev, blockIdx.x, threadIdx.x, collId);
    turn = loadCollCtx(thrdCudaDev, globalCollCtx4Blk7Coll, collId, turn, BASE_CTX_SWITCH_THRESHOLD);
  }

  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d, new blkStatus.currLoadedCollId=%d", thrdCudaDev, blockIdx.x, threadIdx.x, collId, blkStatus.currLoadedCollId);
  return turn;
}

static __device__ int traverseTaskQ(int thrdCudaDev, CollCtx *globalBlk2CollId2CollCtx, int collCount, CQ *cq, CQE *globalCqes, int turn, int *unprogressedCnt, int64_t BASE_CTX_SWITCH_THRESHOLD) {
  int bid = blockIdx.x;

  #if defined(ARRAY_DEBUG) && defined(SHOW_RUNNING_CNT)
    *(blkStatus.barrierCnt + 0 + 11 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    if (blkStatus.numActiveColls == 0) {
      *(blkStatus.barrierCnt + 1 + 11 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
      return turn;
    }
  #else 
    if (blkStatus.numActiveColls == 0) {
      return turn;
    }
  #endif

  // TODO: 循环展开的优化？
  int i = 0;
  for (; i < blkStatus.numActiveColls; i++) {

    // 下边这三个量是不变的。
    int collId = blkStatus.activeCollIds[i];
    int blkLimit = sharedBlkCount4Coll[collId];

    // *(blkStatus.barrierCnt + 0 + 10 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    // *(blkStatus.barrierCnt + 2 + 10 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = collId;

    // 这里不需要再判断blkStatus.collStatus[collId]了，因为这一次循环里只会遍历taskQ一次，出去之后就更新taskQ了。
    if (bid < blkLimit) { // blk天然分化，保留这个条件 // TODO: 如果节省if判断对性能有提升，可以改变处理方法，让所有block处理所有的集合通信。不过好像也省不了。。。总得判断。

      // ***** 先准备好sharedCollCtx，全部线程都参与 *****
      turn = maintainSharedCollCtx(thrdCudaDev, globalBlk2CollId2CollCtx, collId, turn, BASE_CTX_SWITCH_THRESHOLD);

      // *(blkStatus.barrierCnt + 0 + 15 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;

      // ***** 然后调用ofcclFunc *****
      int wid = threadIdx.x / WARP_SIZE;
      if (wid < sharedCollCtx.workElem.header.nWarps) {
        ofcclFuncs[sharedCollCtx.workElem.header.funcIndex](); // 这里边的调用里不涉及__syncthreads().
      }

      // *(blkStatus.barrierCnt + 1 + 15 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;

      // *(blkStatus.barrierCnt + 0 + 13 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
      ofcclBarrier(3); // 跑完一个集合通信，同步一下。
      // *(blkStatus.barrierCnt + 1 + 13 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    }

    // *(blkStatus.barrierCnt + 1 + 10 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  }
  #if defined(ARRAY_DEBUG) && defined(SHOW_RUNNING_CNT)
    *(blkStatus.barrierCnt + 2 + 11 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  #endif

  return turn;
}

// TODO: 考虑在按需启停的场景下，会多次启动，执行上会不会有什么变化。
__global__ void daemonKernel(SQ *sq, CQ *cq, int thrdCudaDev, int collCount, CQE *globalCqes, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx, int *globalVolunteerQuitCounter, int *finallyQuit, BlkStatus *globalBlkStatus, unsigned long long int *barrierCnt, unsigned long long int *collCounters, const int64_t TRAVERSE_TIMES, const int64_t TOLERANT_FAIL_CHECK_SQ_CNT, const int64_t CNT_BEFORE_QUIT, const int64_t TOLERANT_UNPROGRESSED_CNT, const int64_t BASE_CTX_SWITCH_THRESHOLD) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  if (tid == 0) {
    blkStatus.quit = 0;
    blkStatus.iWantToQuit = false;
    blkStatus.seenAllBlockWantToQuitCounter = 0;
    blkStatus.currLoadedCollId = -1;

    sharedCollCtx.saveCtx7Quit = 0;

    #ifdef ARRAY_DEBUG
      blkStatus.barrierCnt = barrierCnt;
      blkStatus.collCounters = collCounters;
    #endif

    BlkStatus *myGlobalBlkStatus = globalBlkStatus + bid;
    blkStatus.hasVolunteerQuitted = myGlobalBlkStatus->hasVolunteerQuitted;

    for (int i = 0; i < collCount; ++i) {
      blkStatus.collStatus[i] = 0;
    }

    if (blkStatus.hasVolunteerQuitted == 0) {
      blkStatus.sqReadFrontier = 0;
      blkStatus.numActiveColls = 0;      

    #ifdef SHOW_SWITCH_CNT
      blkStatus.totalCtxSwitchCnt = 0;
    #endif
    #ifdef SHOW_QUIT_CNT
      blkStatus.totalVolunteerQuitCnt = 0;
      blkStatus.totalUnprogressedQuitCnt = 0;
    #endif
    } else { // 从volunteer quit恢复回来

      blkStatus.numActiveColls = myGlobalBlkStatus->numActiveColls;
      for (int i = 0; i < blkStatus.numActiveColls; ++i) {
        int activeCollId = blkStatus.activeCollIds[i] = myGlobalBlkStatus->activeCollIds[i];
        blkStatus.collStatus[blkStatus.activeCollIds[i]] = 1;
        OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, set blkStatus.collStatus[%d]=1;", thrdCudaDev, blockIdx.x, threadIdx.x, activeCollId);
      }
      
      blkStatus.sqReadFrontier = myGlobalBlkStatus->sqReadFrontier;
      blkStatus.hasVolunteerQuitted = 1;

      #ifdef SHOW_SWITCH_CNT
        blkStatus.totalCtxSwitchCnt = myGlobalBlkStatus->totalCtxSwitchCnt;
      #endif
      #ifdef SHOW_QUIT_CNT
        blkStatus.totalVolunteerQuitCnt = myGlobalBlkStatus->totalVolunteerQuitCnt;
        blkStatus.totalUnprogressedQuitCnt = myGlobalBlkStatus->totalUnprogressedQuitCnt;
      #endif
    }

    if (bid == 0) {
      atomicExch(globalVolunteerQuitCounter, 0); // 稳妥一些，启动的时候，0号block来置零。
    }
  }
  ofcclBarrier(5);

  #ifdef ARRAY_DEBUG
    if (tid == 0) {
      *(blkStatus.barrierCnt + 0 + 5 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    }
  #endif

  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, daemonKernel starts, blkStatus.totalVolunteerQuitCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.totalVolunteerQuitCnt, blkStatus.numActiveColls);
  // OFCCL_LOG_THRD_0(OFCCL_CQE, "Rank<%d> Blk<%d> Thrd<%d>, daemonKernel starts", thrdCudaDev, blockIdx.x, tid);
  // __syncwarp(); // ！！！！！！为了打印log加的！！！！

  // int tempRound = 0;
  int turn = 0;

  turn = initContexts(thrdCudaDev, collCount, globalBlkCount4Coll, globalThrdCount4Coll, globalCollIds, globalDevComm7WorkElems, globalBlk2CollId2CollCtx, turn);

  int checkSQ7TidyTaskQFailCnt = 0;
  int unprogressedCnt = 0;
  while (true) {

    for (int i = 0; i < TRAVERSE_TIMES; i++) {
      if (blkStatus.numActiveColls == 0) {
        break;
      }
      turn = traverseTaskQ(thrdCudaDev, globalBlk2CollId2CollCtx, collCount, cq, globalCqes, turn, &unprogressedCnt, BASE_CTX_SWITCH_THRESHOLD);

      *(blkStatus.barrierCnt + 0 + 17 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
      ofcclBarrier(9);
      *(blkStatus.barrierCnt + 1 + 17 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
      if (tid == 0) { // 遍历完一次之后，当前activeColl的后续工作，
        // 只有完成一个集合通信，才有必要操作taskQ
        int new_numActiveColls = 0;
        for (int i = 0; i < blkStatus.numActiveColls; ++i) {
          int collIdInTaskQ = blkStatus.activeCollIds[i];
          OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d, blkStatus.collStatus is %d", thrdCudaDev, blockIdx.x, threadIdx.x, collIdInTaskQ, blkStatus.collStatus[collIdInTaskQ]);
          if (blkStatus.collStatus[collIdInTaskQ] == -1) { // 不应该有1 的存在了，只有-1或者2
            unprogressedCnt += 1;
            blkStatus.activeCollIds[new_numActiveColls++] = collIdInTaskQ;
          } else if (blkStatus.collStatus[collIdInTaskQ] == 2) {
            unprogressedCnt = 0;

            CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collIdInTaskQ;
            manipulateCQ7ResetDoneColl(thrdCudaDev, collIdInTaskQ, cq, globalCqes, globalCollCtx4Blk7Coll, globalBlk2CollId2CollCtx);
            // 对于完成执行的集合通信应该不用把shmem里的collCtx写回到global mem里边，sendbuff/recvbuff等下次的SQE传过来，剩下的其他都是些静态配置项。
          }
        }
        blkStatus.numActiveColls = new_numActiveColls;
        #ifdef CQE_DEBUG_ALL_RANK
          logTaskQ(1, thrdCudaDev, -1);
        #elif defined(CQE_DEBUG_RANK_X)
          logTaskQ(1, thrdCudaDev, CQE_DEBUG_RANK_X);
        #endif
      }
      *(blkStatus.barrierCnt + 0 + 18 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
      ofcclBarrier(10);
      *(blkStatus.barrierCnt + 1 + 18 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    }

    // *(blkStatus.barrierCnt + 0 + 12 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    ofcclBarrier(6);
    // *(blkStatus.barrierCnt + 1 + 12 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;

    if (tid == 0) {

      // TODO: 可以在这里加一次对globalVolunteerQuitCounter的访问，保证这个小于gridDim.x才进入，但是这样会多一次访存，可能影响性能。同时这也不能100%保证没问题，而且现在的实测还没有出过问题
      //（这只是更稳妥的做法，没啥实际意义，目前没发现这么做能抵御更多的边界情况）
      // volatile int *observeGlobalVolunteerQuitCounterBeforeCheck = globalVolunteerQuitCounter;
      // if (blkStatus.seenAllBlockWantToQuitCounter < 1 && *observeGlobalVolunteerQuitCounterBeforeCheck < gridDim.x) {
    
      if (blkStatus.seenAllBlockWantToQuitCounter < 1) {// 看到过一次大家都要退出，就不再去检查sq了。尽量不产生遗留的block
        checkSQ7TidyTaskQ(thrdCudaDev, sq, globalBlk2CollId2CollCtx, &checkSQ7TidyTaskQFailCnt, finallyQuit, globalVolunteerQuitCounter, &unprogressedCnt);
      }

      // 只有0号线程才会执行checkSQ7TidyTaskQ，自然只有0号线程才会更改checkSQ7TidyTaskQFailCnt，并且进行相应调整。

      // checkSQ7TidyTaskQFailCnt = 0; // 想要结合进oneflow训练，不能禁止主动退出。

      if (checkSQ7TidyTaskQFailCnt > TOLERANT_FAIL_CHECK_SQ_CNT) {
        if (!blkStatus.iWantToQuit) {
          blkStatus.iWantToQuit = true;
          atomicAdd(globalVolunteerQuitCounter, 1); // 我自己第一次想quit的时候，投一票
        }

        volatile int *observeGlobalVolunteerQuitCounter = globalVolunteerQuitCounter; // 为了及时看到大家的投票情况。
        if (*observeGlobalVolunteerQuitCounter == gridDim.x) {
          blkStatus.seenAllBlockWantToQuitCounter++; // 看看大家的投票结果
        } else {
          blkStatus.seenAllBlockWantToQuitCounter = 0;
        }

        // 这样其实还是无法完全保证大家都可以安全退出。好的方法是，让遗留的block也可以退出，或者不产生遗留的block，现在采取的方法是后者。
        if ((blkStatus.seenAllBlockWantToQuitCounter == CNT_BEFORE_QUIT && *observeGlobalVolunteerQuitCounter == gridDim.x)) { // 多次看到大家都要退出这一决议，并且又确认到确实是都要退出了。才会真正执行Volunteer Quit的动作。
          BlkStatus *myGlobalBlkStatus = globalBlkStatus + bid;

          #ifdef SHOW_QUIT_CNT
            ++blkStatus.totalVolunteerQuitCnt;
          #endif
          // 保存blkstatus
          myGlobalBlkStatus->hasVolunteerQuitted = 1;
          blkStatus.quit = 1;

          myGlobalBlkStatus->sqReadFrontier = blkStatus.sqReadFrontier;
          if (blkStatus.numActiveColls > 0) {
            OFCCL_LOG(OFCCL_FATAL, "Rank<%d> Blk<%d> Thrd<%d> blkStatus.numActiveColls = %d, but should be zero", thrdCudaDev, bid, threadIdx.x, blkStatus.numActiveColls);
          }

          #ifdef SHOW_SWITCH_CNT
            myGlobalBlkStatus->totalCtxSwitchCnt = blkStatus.totalCtxSwitchCnt;
          #endif
          #ifdef SHOW_QUIT_CNT
            myGlobalBlkStatus->totalVolunteerQuitCnt = blkStatus.totalVolunteerQuitCnt;
            myGlobalBlkStatus->totalUnprogressedQuitCnt = blkStatus.totalUnprogressedQuitCnt;
          #endif
          // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, Volunteer Quit, checkSQ7TidyTaskQFailCnt = %d, unprogressedCnt = %d, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, checkSQ7TidyTaskQFailCnt, unprogressedCnt, blkStatus.numActiveColls);
        }
      }

      if (unprogressedCnt >= TOLERANT_UNPROGRESSED_CNT && blkStatus.quit != 1) {
        BlkStatus *myGlobalBlkStatus = globalBlkStatus + bid;

        #ifdef SHOW_QUIT_CNT
          ++blkStatus.totalVolunteerQuitCnt;
          ++blkStatus.totalUnprogressedQuitCnt;
        #endif

        // 保存blkstatus
        myGlobalBlkStatus->hasVolunteerQuitted = 1;
        blkStatus.quit = 1;
        myGlobalBlkStatus->sqReadFrontier = blkStatus.sqReadFrontier;

        myGlobalBlkStatus->numActiveColls = blkStatus.numActiveColls;
        for (int i = 0; i < blkStatus.numActiveColls; ++i) {
          myGlobalBlkStatus->activeCollIds[i] = blkStatus.activeCollIds[i];
        }

        #ifdef SHOW_SWITCH_CNT
          myGlobalBlkStatus->totalCtxSwitchCnt = blkStatus.totalCtxSwitchCnt;
        #endif
        #ifdef SHOW_QUIT_CNT
          myGlobalBlkStatus->totalVolunteerQuitCnt = blkStatus.totalVolunteerQuitCnt;
          myGlobalBlkStatus->totalUnprogressedQuitCnt = blkStatus.totalUnprogressedQuitCnt;
        #endif
      }
    }

    // *(blkStatus.barrierCnt + 0 + 9 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    ofcclBarrier(7); // prims_simple里用的是8和15。
    // *(blkStatus.barrierCnt + 1 + 9 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;

    // // daemonKernel一开始这个数组用不上，可以用来记点其他信息

    #ifdef ARRAY_DEBUG
    #ifdef SHOW_RUNNING_CNT
      if (tid == 0) {
        *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 33 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.totalCtxSwitchCnt;
        *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 34 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
        *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 35 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = unprogressedCnt;
      }
    #endif
    #endif

    // 记录数组的前10项，未必都是有效的。所有线程都做，看到的应该是一样的。
    // for (int i = 0; i < PrintTestQNum; i++) {
    //   *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + (36 + i) * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.activeCollIds[i];
    // }

    if (blkStatus.quit == 1) {
      #ifdef SHOW_QUIT_CNT
        if (*finallyQuit == 1) {
          OFCCL_LOG_THRD_0(OFCCL_FINAL_OR_VOLUNTEER_QUIT, "Rank<%d> Blk<%d> Thrd<%d> collCount=%d, totalCtxSwitchCnt=%llu, totalVolunteerQuitCnt=%llu, totalUnprogressedQuitCnt=%llu", thrdCudaDev, bid, tid, collCount, blkStatus.totalCtxSwitchCnt, blkStatus.totalVolunteerQuitCnt, blkStatus.totalUnprogressedQuitCnt);
        }
      #endif
      // OFCCL_LOG_THRD_0(OFCCL_CQE, "Rank<%d> Blk<%d> Thrd<%d>, daemonKernel quits", thrdCudaDev, blockIdx.x, tid);
      #ifdef ARRAY_DEBUG
        if (tid == 0) {
          *(blkStatus.barrierCnt + 1 + 5 * BARCNT_INNER_SIZE + threadIdx.x * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
          #ifdef SHOW_QUIT_CNT
            *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 65 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.totalVolunteerQuitCnt;
            *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 66 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.totalUnprogressedQuitCnt;
          #endif
        }
      #endif
      return;
    }
  }
}