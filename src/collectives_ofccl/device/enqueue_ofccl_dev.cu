#include "hip/hip_runtime.h"
#include "enqueue_ofccl_dev.h"

// TODO: nccl最新的代码里，这部分的设计和实现都变了。
// 
// Copy src to dst and fill extra size with zeroes
// 这个是保证在一次调用复制完最多512B，并且以16B为单位。
// 这个不要求src dst同一类型
template<typename Tdst, typename Tsrc>
static __device__ void copyToShmemOneShot(Tdst *dst, Tsrc const *src, int tid, int nthreads) { // nccl的这个的函数签名里有个nthreads参数，但是并没有用，应该是为了和下边那个作区分，现在我们可以区分开了，反而带上nthreads是区分不开的。
  static_assert(sizeof(Tdst)%(2*sizeof(uint64_t )) == 0 && sizeof(Tsrc)%(2*sizeof(uint64_t)) == 0,
      "copyToShmemOneShot needs sizes which are multiple of 16B");
  static_assert(sizeof(Tdst) >= sizeof(Tsrc), "Tdst size is too small");
  static_assert(sizeof(Tdst) <= WARP_SIZE*2*sizeof(uint64_t), "copyToShmemOneShot limited to 512B to make sure it can always be done in one cycle");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  uint64_t *shmemPtr = shmemCvtPtr_ofccl(d); // 由于这个地方，这个函数只能用于dst是shmem的情况了。
  int offset = 2*tid;
  uint64_t v0, v1;
  if (offset >= sizeof(Tsrc)/sizeof(uint64_t)) {
    v0 = v1 = 0ULL;
  } else {
    v0 = s[offset] ; v1 = s[offset+1];
  }
  if (offset < sizeof(Tdst)/sizeof(uint64_t)) storeShmem128_ofccl(shmemPtr+offset, v0, v1);
}

// 这个可以直接用到任意一轮搞不完的数据结构的复制吧。
// 这个要求src dst同一类型。
// turn的作用：   
template<typename T>
static __device__ int copyToShmemLoop(T *dst, T const *src, int tid, int nthreads, int turn=0) {
  static_assert(sizeof(uint64_t) <= alignof(T), "Uhoh");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  int t = tid - turn;
  if (t < 0) t += nthreads;
  int n = sizeof(T)/sizeof(uint64_t); // n 代表要复制的数据结构包含了几个8Byte

  int delta = (n + WARP_SIZE-1) & -WARP_SIZE; // round up to warp lane 0; 要把n和WARP_SIZE处理对齐了。
  //  32 = 0000 0000 0010 0000
  // -32 = 1111 1111 1110 0000，低位不变，高位都置1。大一的东西忘却了。。
  // 所以delta相当于n相对于32的“向上取整”，即向上取到32的整数倍。

  if (delta < nthreads) { // 总的要传的 8Byte 的个数小于blockDim.x（我们的case里是thrdLimit）
    turn += delta;
    if (turn >= nthreads) turn -= nthreads; // 在第一次调用里这个不会成立，应该是为了后续的调用使用
  }
  else
    turn = 0; // 如果总的要传的 8Byte 的个数超过了blockDim.x，那就不用管turn了。所以turn就是为了雨露均沾，让所有线程都干活

  n -= t; // 对每个线程来说，砍掉比tid小的几项，不用自己管。
  d += t; // 对每个线程来说，自己从tid的偏移量开始管。
  s += t;
  #pragma unroll // 指示要循环展开。
  for (int i=0; i < divUp(sizeof(T), WARP_SIZE*sizeof(uint64_t)); i++) {
    if (n > 0) {
      *d = *s;
      d += nthreads;
      s += nthreads;
      n -= nthreads; // “一轮”完成 nthreads个8 Byte的复制。
    }
  }
  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> hello", sharedCollCtx.rank, blockIdx.x, tid);
  return turn;
}

// 这个的目的应该是在“切片并行复制”之后，恢复标量的语义
// 但是没用，而且在buffer里的数据是0.5，或者其他数字时，导致卡住。log发现buffer里的数字是0.25，可以正常运行，并且没有进入这里。所以直接注释了吧。
// TODO: 但是这里卡住，总还是怪怪的。有空看看吧。
// static __device__ void ofcclRedopPtrDeref(struct ncclWorkElem* we) {
//   if (we->header.type != ncclWorkTypeUnused && we->redOpArgIsPtr) {
//     /* redOpArg is a pointer to the scalar value, so we'll dereference it
//      * here so that redOpArg holds the bits of the scalar going forward.
//      * The tricky thing is we don't know its type T since that's encoded in
//      * the funcIndex. Because it would be difficult to get sizeof(T) from
//      * funcIndex, we'll cheat and just dereference the largest possible size
//      * given the alignment of the pointer. We might be reading in more bytes
//      * than we need but that's harmless.
//      */
//     if (we->redOpArg%2 != 0)
//       we->redOpArg = *reinterpret_cast<uint8_t*>(we->redOpArg);
//     else if (we->redOpArg%4 != 0)
//       we->redOpArg = *reinterpret_cast<uint16_t*>(we->redOpArg);
//     else if (we->redOpArg%8 != 0)
//       we->redOpArg = *reinterpret_cast<uint32_t*>(we->redOpArg);
//     else
//       we->redOpArg = *reinterpret_cast<uint64_t*>(we->redOpArg);
//     // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, we->redOpArgIsPtr = %d, we->redOpArg = %llu", sharedCollCtx.rank, blockIdx.x, threadIdx.x, we->redOpArgIsPtr, we->redOpArg);
//   }
// }

// share mem用超了。
// TODO: 可以不同的algo、proto使用不同的数据类型，不过可以看看是不是有意义
__shared__ CollCtx sharedCollCtx; // 不能static，primitives要用

__shared__ BlkStatus blkStatus; // 取消static，放到prim里边打印log。
// TODO: 下边这几个可以尝试用constant，先不急
static __shared__ int sharedCollIds[MAX_LENGTH]; // prepareColl会接受用户传进来的collId，而prepareColl工作在每个rank上，我们不能假设各个rank会收到连续的collId，所以用一个数组把收到的collId整理起来，其实相当于是维护了一个map，但是cuda上没有map，只好用这种方式
static __shared__ int sharedBlkCount4Coll[MAX_LENGTH];
static __shared__ int sharedThrdCount4Coll[MAX_LENGTH];

static __device__ int sqRead(SQ *sq, unsigned long long int sqReadFrontier, SQE *target, int thrdCudaDev) {
  int bid = blockIdx.x;
  int sqeCollId;
  
  // int tid = threadIdx.x;
  // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> enter sqRead, sqHead=%llu, sqTail=%llu, empty=%d, RingBuffer_get(sq, sqReadFrontier)->counter=%d, RingBuffer_get(sq, sqReadFrontier)->collId=%d, RingBuffer_get(sq, sqReadFrontier)->quit=%d, RingBuffer_get(sq, sqReadFrontier)->logicHead=%d, GetLogicFrontier(sq, sqReadFrontier)=%llu", thrdCudaDev, bid, tid, RingBuffer_logic_head(sq), RingBuffer_logic_tail(sq), RingBuffer_empty(sq), RingBuffer_get(sq, sqReadFrontier)->counter, RingBuffer_get(sq, sqReadFrontier)->collId, RingBuffer_get(sq, sqReadFrontier)->quit, RingBuffer_get(sq, sqReadFrontier)->logicHead, GetLogicFrontier(sq, sqReadFrontier));
  if (RingBuffer_empty(sq)) {
    return -1;
  }
  // 先读过来，然后再判断，最后更新状态：sqe->counter; 以及在恰当的时候commit read
  *target = *RingBuffer_get(sq, sqReadFrontier);
  if (target->quit) {
    // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Get quit", thrdCudaDev, bid, tid);
    return 0;
  }

  // 先判断一下相应的collId是不是该自己的bid处理，不该自己处理直接返回-1
  sqeCollId = target->collId;
  // OFCCL_LOG(OFCCL, "Blk<%d>, Thrd<%d> sharedBlkCount4Coll[%d]=%d", thrdCudaDev, bid, tid, sqeCollId, sharedBlkCount4Coll[sqeCollId]);
  if (bid >= sharedBlkCount4Coll[sqeCollId]) {
    return -1;
  } else {
    // 自己读到之后，更新相应的counter；至于读到的sqe对应的collId是不是该自己处理，是caller的事。
    // 如果发现自己读完之后，所有block都读了，那么commit read
    // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> PREPARE to increase counter(curr=%d) for sqe of coll_id = %d", thrdCudaDev, bid, tid, RingBuffer_get(sq, sqReadFrontier)->counter, sqeCollId);
    int old_counter = atomicAdd(&(RingBuffer_get(sq, sqReadFrontier)->counter), 1);
    __threadfence_system();
    // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> increase counter to %d for sqe of coll_id = %d", thrdCudaDev, bid, tid, old_counter + 1, sqeCollId);
    
    if (old_counter + 1 == sharedBlkCount4Coll[sqeCollId]) {
      
      unsigned long long int old_head = atomicAdd(&sq->head, 1);

      __threadfence_system();
      // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sqe of coll_id = %d commit read, new sqHead is %llu", thrdCudaDev, bid, tid, sqeCollId, old_head + 1);
    }
  }
  
  return 0;
}

static __device__ int cqWrite(CQ *cq, CQE *cqe, int thrdCudaDev) {
  if (RingBuffer_full(cq)) {
    // not an error; caller keeps trying.
    return -1;
  }

  *RingBuffer_get_tail(cq) = *cqe;

  __threadfence_system();

  atomicAdd(&cq->tail, 1); // uint64, 一往无前++
  // RingBuffer_commit_write(cq, 1);

  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> cqWrite done, RingBuffer_full(cq)=%d, cqHead=%llu, cqTail=%llu", thrdCudaDev, blockIdx.x, threadIdx.x, RingBuffer_full(cq), RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
  // OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> cqWrite done, RingBuffer_full(cq)=%d, cqHead=%llu, cqTail=%llu", thrdCudaDev, blockIdx.x, threadIdx.x, RingBuffer_full(cq), RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));

  return 0;
}

// TODO: 为了性能，考虑恢复成多线程一起复制的写法。
static __device__ void copyNcclWorkElem (struct ncclWorkElem &dstElem, const struct ncclWorkElem &srcElem) {
  dstElem.header.funcIndex = srcElem.header.funcIndex;
  dstElem.header.type = srcElem.header.type;
  dstElem.header.nWarps = srcElem.header.nWarps;
  dstElem.header.isLast = srcElem.header.isLast;

  dstElem.regUsed = srcElem.regUsed;
  dstElem.direct = srcElem.direct;
  dstElem.redOpArgIsPtr = srcElem.redOpArgIsPtr;
  dstElem.sendbuff = srcElem.sendbuff;
  dstElem.recvbuff = srcElem.recvbuff;
  dstElem.count = srcElem.count;
  dstElem.lastChunkSize = srcElem.lastChunkSize;
  dstElem.root = srcElem.root;
  dstElem.bid = srcElem.bid;
  dstElem.nChannels = srcElem.nChannels;
  dstElem.redOpArg = srcElem.redOpArg;
}

static __device__ int initContexts(int thrdCudaDev, int collCount, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx, int turn) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  // int nthreads = blockDim.x;
  // 构建任务列表
  if (tid == 0) {
    for (int i = 0; i < collCount; i++) {
      int collId = sharedCollIds[i] = globalCollIds[i];
      // 以下这两个变量会限制很多行为。
      int blkLimit = sharedBlkCount4Coll[collId] = globalBlkCount4Coll[collId];
      sharedThrdCount4Coll[collId] = globalThrdCount4Coll[collId];

      // 下边这部分逻辑在在blkStatus.hasVolunteerQuitted == 1的情况下不执行，曾经退出过，恢复的时候就不要重新初始化了。
      if (blkStatus.hasVolunteerQuitted == 0) {
        // 每个block一份globalShmem
        CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;

        // ***** 移植ncclKernel的逻辑 *****
        if (bid < blkLimit) {
          // ncclDevComm *comm = globalDevComm7WorkElems[collId].comm;
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->comm), comm, tid, nthreads, turn);
          // // 一个奇技淫巧：get address of channel without incurring indirect load from ncclDevComm::channels
          // // 这里通过bid选择了合适的channel，很多集合通信真正执行时用到的硬件信息就存在channel里边。
          // ncclChannel *channel = &((ncclDevCommAndChannels*)comm)->channels[bid];
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->channel), channel, tid, nthreads, turn); // 尝试使用oneshot，会报错warp misaligned，所以看来必须用loop。

          // // nccl中限制只在bid=0里进行这样的拷贝，对于ofccl而言，ofcclShmem就是任务列表，所以对于所有的线程，我们都把同样的work存进去；
          // turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->work.elems[0]), &(globalDevComm7WorkElems[collId].first), tid, nthreads, turn); // nccl 2.12里边这地方用copyToShmemOneShot进行拷贝，但是oneShot的实现使用了与shared mem相关的内联汇编，所以这里也使用loop进行拷贝。
          // // nccl中接下来要处理channel.workFifoDev，然而对于目前的ofccl，只处理first就好，channel.workFifoDev不会有其他任务了。
          // __syncthreads(); // 等待全部线程加载完成

          // if (globalCollCtx4Blk7Coll->work.header.type == ncclWorkTypeColl) {
          //   // #define NCCL_MAX_WORK_ELEMENTS (NCCL_WORK_SIZE / sizeof(struct ncclWorkElem))=512/64=8
          //   // 原来这个写法，应该是想修改we->redOpArg，不过修改we->redOpArg一个线程就够了，所以让理论上最多的线程来工作，咱们保留就好。
          //   if (tid < NCCL_MAX_WORK_ELEMENTS) ofcclRedopPtrDeref(&(globalCollCtx4Blk7Coll->work.elems[tid]));
          // } // 目前不用考虑其他ncclWorkType
          // __syncthreads();
        
          /* ****** 手动加载用得到的shmemData ****** */
          ncclDevComm *comm = globalDevComm7WorkElems[collId].comm;
          ncclChannel *channel = &((ncclDevCommAndChannels*)comm)->channels[bid];

          globalCollCtx4Blk7Coll->ringPrev = channel->ring.prev;
          globalCollCtx4Blk7Coll->ringNext = channel->ring.next;
          globalCollCtx4Blk7Coll->ringIndex = channel->ring.index;
          globalCollCtx4Blk7Coll->devPeers = channel->devPeers; // 直接赋值指针

          globalCollCtx4Blk7Coll->rank = comm->rank;
          globalCollCtx4Blk7Coll->nRanks = comm->nRanks;
          globalCollCtx4Blk7Coll->abortFlag = comm->abortFlag;

          for (int i = 0; i < NCCL_NUM_PROTOCOLS; i++) {
            globalCollCtx4Blk7Coll->buffSizes[i] = comm->buffSizes[i];
          }

          copyNcclWorkElem(globalCollCtx4Blk7Coll->workElem, globalDevComm7WorkElems[collId].first);

          /* ****** 上下文 ****** */
          globalCollCtx4Blk7Coll->executing = 0;
          // globalCollCtx4Blk7Coll->numDoneThrds = 0;

          globalCollCtx4Blk7Coll->sqeReadCnt = 0;
          globalCollCtx4Blk7Coll->cqeWriteCnt = 0;
          
          // bugfix: 下边原来都是设置的globalBlk2CollId2CollCtx->XXXX，相当于都设置了第0个block的第0个coll。。。。。。。
          globalCollCtx4Blk7Coll->saveCtx7Quit = 0;
          globalCollCtx4Blk7Coll->loadAgain = 0;
          globalCollCtx4Blk7Coll->slice4SimpleGenericOp = 0;
          globalCollCtx4Blk7Coll->offset4SimpleGenericOp = 0;

          globalCollCtx4Blk7Coll->currentStep4RingAllReduce = 0;
          globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = 0;
        }
      }
    }
  }
  *(blkStatus.barrierCnt + 0 + 5 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  // __syncthreads(); // 保证全部线程看得到更新的内容
  ofcclBarrier(1);
  *(blkStatus.barrierCnt + 1 + 5 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  // OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, after initContexts, blkStatus.totalVolunteerQuitCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.totalVolunteerQuitCnt, blkStatus.numActiveColls);
  return turn;
}

// 为了初步实现按需启停，增加一个“空read计数，读不到新的，增加计数”
static __device__ void checkSQ(int thrdCudaDev, SQ *sq, CollCtx *globalBlk2CollId2CollCtx, int *failCnt, int *finallyQuit) {
  int bid = blockIdx.x;
  // int tempThrdCudaDev = thrdCudaDev;
  
  SQE target;
  // TODO: 感觉没必要，这里是要读，而fence应该是写者那边加的。
  // __threadfence_system(); // make sure read new head and tail.

  // OFCCL_LOG_BLK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, sq @ %p", thrdCudaDev, bid, threadIdx.x, sq);
  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, sq->head = %llu, sq->tail = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, bid, threadIdx.x, sq->head, sq->tail, blkStatus.numActiveColls);

  if (blkStatus.sqReadFrontier < sq->head) {
    // 如果当前bid比较大，一些SQE不需要这个block处理，就会跳过。导致当前block的frontier小于head。
    // 不给sqRead增加返回值种类；否则会增加无谓的sqRead调用、增加访存次数。

    // !! 与volunteer quit相关，对于编号较大的block来说，这代表其他block在忙，自己不该退出。
    // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> my blkStatus.sqReadFrontier = %llu, less than sq->head = %llu", thrdCudaDev, bid, threadIdx.x, GetLogicFrontier(sq, blkStatus.sqReadFrontier), RingBuffer_logic_head(sq));

    blkStatus.sqReadFrontier = sq->head;
    *failCnt = 0;
  }

  // 能读到，假如是正常SQE，把信息在任务列表里记录一下；假如是quit，那也记录一下
  // 读不到新东西那就算了
  if (RingBuffer_logic_tail(sq) == GetLogicFrontier(sq, blkStatus.sqReadFrontier) || sqRead(sq, blkStatus.sqReadFrontier, &target, thrdCudaDev) == -1) {
    *failCnt += 1;
    if (blkStatus.numActiveColls > 0) {
      *failCnt = 0;
    }
    return;
  } else {
    // TODO: 更改failCnt的更新逻辑，觉得自己死锁了，虽然任务列表不空，但是半天动不了，也可以退。
    *failCnt = 0;
    blkStatus.sqReadFrontier++;
    if (target.quit) {
      blkStatus.quit = 1;
      // if (bid == 0) {
        *finallyQuit = 1; // TODO: 为了最后每个block都保证打印统计信息，挺不优雅的
      // }
      // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Main Thrd of Blk quit, *finallyQuit = %d", thrdCudaDev, bid, threadIdx.x, *finallyQuit);
      // OFCCL_LOG_BLK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Main Thrd of Blk quit, *finallyQuit = %d", thrdCudaDev, bid, threadIdx.x, *finallyQuit);
      return;
    }

    // 正常读到了SQE的话，需要往global的globalBlk2CollId2CollCtx表项里边写入，更新blkStatus.numActiveColls
    int newActiveCollId = target.collId;
    int blkLimit = sharedBlkCount4Coll[newActiveCollId];
    if (bid < blkLimit) {
      CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + newActiveCollId;
      if (globalCollCtx4Blk7Coll->executing == 1) {
        OFCCL_LOG(OFCCL_FATAL, "Rank<%d> Blk<%d> Thrd<%d> globalCollCtx4Blk7Coll->executing should be 0! sq->head = %llu, sq->tail = %llu, blkStatus.sqReadFrontier = %llu", thrdCudaDev, bid, threadIdx.x, RingBuffer_logic_head(sq), RingBuffer_logic_tail(sq), GetLogicFrontier(sq, blkStatus.sqReadFrontier));
      }
      // TODO: 可以考虑一下这个地方加入原子操作，保证没有重入的风险。重入指一个正在执行的集合通信又被提起请求。
      // 虽然这里是操作globalMemory，但是我们设计的是各个block自己的数据结构自己操作。具体操作的都是每个block的0号线程，所以应该不会有啥问题。
      globalCollCtx4Blk7Coll->executing = 1;
      globalCollCtx4Blk7Coll->workElem.sendbuff = target.sendbuff;
      globalCollCtx4Blk7Coll->workElem.recvbuff = target.recvbuff;
      
      /* IF_CHECK checkSQ里收到sqe的时候浅打印一下初始sendbuff。作用一般。*/

      // float *sendptr = (float *)target.sendbuff;
      // for (int i = 0; i < buffPrintNum; i++) {
      //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sendbuff @%p sendbuff[%d]=%f", thrdCudaDev, bid, threadIdx.x, target.sendbuff, i, *(sendptr + i));
      // }
      // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> recvbuff @%p", thrdCudaDev, bid, threadIdx.x, target.recvbuff);

      // block的0号线程操作shmem，不用原子操作
      blkStatus.numActiveColls += 1;
      // __threadfence_block(); // 应该不需要，返回后外边有个barrier

      // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> get coll_id = %d, blkStatus.sqReadFrontier updates to %llu, blkStatus.numActiveColls = %d, globalCollCtx4Blk7Coll->sqeReadCnt = %llu", thrdCudaDev, bid, threadIdx.x, target.collId, GetLogicFrontier(sq, blkStatus.sqReadFrontier), blkStatus.numActiveColls, globalCollCtx4Blk7Coll->sqeReadCnt++);
    }
  }
}

static __device__ int loadCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int collId, int turn) {
  int tid = threadIdx.x;
  // int nthreads = blockDim.x;

  // turn = copyToShmemLoop(&sharedCollCtx.comm, &(globalCollCtx4Blk7Coll->comm), tid, nthreads, turn);
  // turn = copyToShmemLoop(&sharedCollCtx.channel, &(globalCollCtx4Blk7Coll->channel), tid, nthreads, turn);
  // // copyToShmemOneShot(&sharedCollCtx.work, &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads); // TODO: 用了这个会报错misaligned，就先loop吧
  // turn = copyToShmemLoop(&(sharedCollCtx.work.elems[0]), &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads, turn);
  // sharedCollCtx.work.elems[0].header.nWarps = globalCollCtx4Blk7Coll->work.elems[0].header.nWarps;
  // // turn = copyToShmemLoop(&sharedCollCtx.work, &(globalCollCtx4Blk7Coll->work), tid, nthreads, turn);
  // __syncthreads(); // 全部线程都执行，可以使用这个同步。

  if (tid == 0) {
    sharedCollCtx.ringPrev = globalCollCtx4Blk7Coll->ringPrev;
    sharedCollCtx.ringNext = globalCollCtx4Blk7Coll->ringNext;
    sharedCollCtx.ringIndex = globalCollCtx4Blk7Coll->ringIndex;
    sharedCollCtx.devPeers = globalCollCtx4Blk7Coll->devPeers;

    sharedCollCtx.rank = globalCollCtx4Blk7Coll->rank;
    sharedCollCtx.nRanks = globalCollCtx4Blk7Coll->nRanks;
    sharedCollCtx.abortFlag = globalCollCtx4Blk7Coll->abortFlag;

    for (int i = 0; i < NCCL_NUM_PROTOCOLS; i++) {
      sharedCollCtx.buffSizes[i] = globalCollCtx4Blk7Coll->buffSizes[i];
    }

    copyNcclWorkElem(sharedCollCtx.workElem, globalCollCtx4Blk7Coll->workElem);

    // // for debug
    // {
    //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
    //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
    //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
    //   uint64_t head = recvConn->step;
    //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
    //   uint64_t tail = sendConn->step;
    //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d load head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, tid, collId, head, tail);
    // }

    // TODO: 目前只有simple ring allreduce，之后考虑通用性和扩展性。
    // 加载algo、proto、func相关的运行上下文。

    // sharedCollCtx.saveCtx7Quit = globalCollCtx4Blk7Coll->saveCtx7Quit; // 这个看起来也可以充当标记是否是跑了一半的标记位
    sharedCollCtx.saveCtx7Quit = 0; // 每次加载的时候，重置。
    sharedCollCtx.loadAgain = globalCollCtx4Blk7Coll->loadAgain;
    sharedCollCtx.slice4SimpleGenericOp = globalCollCtx4Blk7Coll->slice4SimpleGenericOp;
    sharedCollCtx.offset4SimpleGenericOp = globalCollCtx4Blk7Coll->offset4SimpleGenericOp;

    // sharedCollCtx.totalSteps4RingAllReduce = 2 * sharedCollCtx.nRanks - 1;
    sharedCollCtx.currentStep4RingAllReduce = globalCollCtx4Blk7Coll->currentStep4RingAllReduce;
    sharedCollCtx.gridOffset4RingAllReduce = globalCollCtx4Blk7Coll->gridOffset4RingAllReduce;
    // __threadfence_block();
  }
  *(blkStatus.barrierCnt + 0 + 6 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  // __syncthreads(); // TODO: 这个同步看来不能换成syncwarp，会报错cudaErrorIllegalInstruction。
  // __syncwarp();
  ofcclBarrier(1);

  *(blkStatus.barrierCnt + 1 + 6 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
 
  return turn;
}

static __device__ void manipulateCQ7ResetDoneColl(int thrdCudaDev, int doneCollId, CQ *cq, CQE *globalCqes, CollCtx *globalCollCtx4Blk7Coll) {
  // 协调所有blk，发现所有blk都完成，最后一个blk发送CQE
  int old_counter = atomicAdd(&(globalCqes[doneCollId].counter), 1);
  __threadfence(); // cqes在global memory里边，全部block关心。

  if (old_counter + 1 == sharedBlkCount4Coll[doneCollId]) {
    atomicExch(&globalCqes[doneCollId].counter, 0);
    while (cqWrite(cq, globalCqes + doneCollId, thrdCudaDev) == -1) {
      // tempRound++;
      // if(tempRound % tempPrintRound == 0) {
      //   OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> cqWrite fail, RingBuffer_full(cq)=%d, cqHead=%llu, cqTail=%llu", thrdCudaDev, blockIdx.x, tid, RingBuffer_full(cq), RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
      // }

    }
    // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> cqe coll_id = %d, cqeWriteCnt = %llu", thrdCudaDev, blockIdx.x, tid, doneCollId, globalCollCtx4Blk7Coll->cqeWriteCnt++);
    
    __threadfence();
  }

  
  blkStatus.numActiveColls -= 1;
  blkStatus.currActiveCollId = -1;

  globalCollCtx4Blk7Coll->executing = 0;
  globalCollCtx4Blk7Coll->loadAgain = 0;
  globalCollCtx4Blk7Coll->saveCtx7Quit = 0;

  // 需要把上下文也重置了，不然多次运行会有问题。
  globalCollCtx4Blk7Coll->slice4SimpleGenericOp = 0;
  globalCollCtx4Blk7Coll->offset4SimpleGenericOp = 0;
  globalCollCtx4Blk7Coll->currentStep4RingAllReduce = 0;
  globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = 0;

  // for debug
  // {
  //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
  //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
  //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
  //   uint64_t head = recvConn->step;
  //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
  //   uint64_t tail = sendConn->step;
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d done head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, tid, doneCollId, head, tail);
  // }

  
  // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> update CQE for coll_id = %d, blkStatus.numActiveColls = %d globalCollCtx4Blk7Coll->cqeWriteCnt = %llu", thrdCudaDev, blockIdx.x, threadIdx.x, doneCollId, blkStatus.numActiveColls, globalCollCtx4Blk7Coll->cqeWriteCnt++);

  /* IF_CHECK 如果要检查对错，把下边露出来 */

  // float *sendptr = (float *)sharedCollCtx.work.elems[0].sendbuff;
  // float *ptr = (float *)sharedCollCtx.work.elems[0].recvbuff;
  // // for (int i = 0; i < 0+buffPrintNum; i++) {
  // //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sendbuff @ %p sendbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].sendbuff, i, *(sendptr + i));
  // //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> recvbuff @ %p recvbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].recvbuff, i, *(ptr + i));
  // // }
  // for (int i = buffPrintStart; i < buffPrintStart+buffPrintNum; i++) {
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sendbuff @ %p sendbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].sendbuff, i, *(sendptr + i));
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> recvbuff @ %p recvbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].recvbuff, i, *(ptr + i));
  // }

}

static __device__ void saveExcutingCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int thrdLimit, int collId) {
  // globalCollCtx4Blk7Coll->saveCtx7Quit = sharedCollCtx.saveCtx7Quit;
  globalCollCtx4Blk7Coll->loadAgain = sharedCollCtx.loadAgain;
  globalCollCtx4Blk7Coll->slice4SimpleGenericOp = sharedCollCtx.slice4SimpleGenericOp;
  globalCollCtx4Blk7Coll->offset4SimpleGenericOp = sharedCollCtx.offset4SimpleGenericOp;

  globalCollCtx4Blk7Coll->currentStep4RingAllReduce = sharedCollCtx.currentStep4RingAllReduce;
  globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = sharedCollCtx.gridOffset4RingAllReduce;

  blkStatus.totalCtxSwitchCnt++;
  blkStatus.currActiveCollId = -1;
  
  // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, blkStatus.totalCtxSwitchCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.totalCtxSwitchCnt, blkStatus.numActiveColls);
  
  // for debug
  // {
  //   struct ncclPeer *recvPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringPrev];
  //   struct ncclPeer *sendPeer = &sharedCollCtx.devPeers[sharedCollCtx.ringNext];
  //   struct ncclConnInfo *recvConn = &recvPeer->recv[0].conn;
  //   uint64_t head = recvConn->step;
  //   struct ncclConnInfo *sendConn = &sendPeer->send[0].conn;
  //   uint64_t tail = sendConn->step;
  //   OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d save head = %llu, tail = %llu", sharedCollCtx.rank, blockIdx.x, tid, collId, head, tail);
  // }
}

static __device__ int traverseGlobalCollCtx(int thrdCudaDev, CollCtx *globalBlk2CollId2CollCtx, int collCount, CQ *cq, CQE *globalCqes, int turn) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  int numSeenActiveColls = 0; // 用这个和blkStatus.numActiveColls 配合，减少下边的循环次数，在block天然分化的前提下，看起来可以直接用。

  if (blkStatus.numActiveColls == 0) {
    *(blkStatus.barrierCnt + 1 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
    *(blkStatus.barrierCnt + 2 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = collCount;
    return turn;
  }

  int currNumActiveColls = blkStatus.numActiveColls;

  int i = 0;
  for (; i < collCount; i++) {
    ofcclBarrier(1); // TODO: debug经验。发现读shmem不一致，那就在读之前稳稳加一个barrier
    *(blkStatus.barrierCnt + 0 + 10 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = numSeenActiveColls;
    *(blkStatus.barrierCnt + 1 + 10 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
    *(blkStatus.barrierCnt + 2 + 10 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;
    ofcclBarrier(1);

    // TODO: 先取消这个减少循环次数的优化试试能不能解决bug。
    if (numSeenActiveColls >= currNumActiveColls) { // !!! bug fix: blkStatus.numActiveColls会在下边完成一个coll之后被减小，而我们这里控制循环次数，是希望使用一个固定的值。
      *(blkStatus.barrierCnt + 0 + 15 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = i;
      *(blkStatus.barrierCnt + 1 + 15 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = numSeenActiveColls;
      *(blkStatus.barrierCnt + 2 + 15 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
      *(blkStatus.barrierCnt + 3 + 15 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;
      break;
    }

    // 下边这三个量是不变的。
    int collId = sharedCollIds[i];
    int blkLimit = sharedBlkCount4Coll[collId];
    int thrdLimit = sharedThrdCount4Coll[collId];

    if (bid < blkLimit) { // blk天然分化，保留这个条件 // TODO: 如果节省if判断对性能有提升，可以改变处理方法，让所有block处理所有的集合通信。不过好像也省不了。。。总得判断。
      // block内全部线程都执行：
      CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;
      if (globalCollCtx4Blk7Coll->executing == 1) {
        if (tid == 0) { // TODO: 主要是打log用的，不打log可以删掉，省一个if。
          blkStatus.currActiveCollId = collId; // 0号线程修改shmem，应该不用原子操作。
          // __threadfence_block(); // 主要这里有个fence操作，所以可能会带来分化的危险。
        }
        // __syncthreads(); // 没必要为了这么个玩意浪费同步。这里同步线程，那上边就不用插入fence了。

        numSeenActiveColls++;
        
        // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, before loadCollCtx for coll_id = %d blkStatus.numActiveColls = %d, numSeenActiveColls = %d", thrdCudaDev, bid, tid, collId, blkStatus.numActiveColls, numSeenActiveColls);
        // __syncwarp(); // ！！！！！！为了打印log加的！！！！

        // if (tid == 0) {
        //   OFCCL_LOG1(OFCCL, "hi");
        // }

        // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, hi", thrdCudaDev, bid, tid);

        // ***** 先准备好sharedCollCtx，全部线程都参与 *****
        // 这个load事实上也只应该影响工作的warp，不过由于是操作shmem，所以其他warp没办法，也会受影响。
        turn = loadCollCtx(thrdCudaDev, globalCollCtx4Blk7Coll, collId, turn); // 只load一个到shmem

        // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, after loadCollCtx for coll_id = %d, sharedCollCtx.workElem.header.nWarps = %u, sharedCollCtx.workElem.sendbuff @ %p, sharedCollCtx.workElem.recvbuff @ %p", thrdCudaDev, bid, tid, collId, sharedCollCtx.workElem.header.nWarps, sharedCollCtx.workElem.sendbuff, sharedCollCtx.workElem.recvbuff);
        // __syncwarp(); // ！！！！！！为了打印log加的！！！！
        
        // ***** 然后调用ofcclFunc *****

        // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, before ofcclFuncs[%d], sharedCollCtx.saveCtx7Quit = %d, coll_id = %d", thrdCudaDev, bid, tid, sharedCollCtx.workElem.header.funcIndex, sharedCollCtx.saveCtx7Quit, collId);
        // __syncwarp();  // ！！！！！！为了打印log加的！！！
        int wid = threadIdx.x / WARP_SIZE;
        if (wid < sharedCollCtx.workElem.header.nWarps) {
          ofcclFuncs[sharedCollCtx.workElem.header.funcIndex](); // 这里边的调用里不涉及__syncthreads().
          // 根据sharedCollCtx.saveCtx7Quit的情况进行不同处理。
          // OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, ofcclFuncs[%d]() return", thrdCudaDev, blockIdx.x, threadIdx.x, sharedCollCtx.workElem.header.funcIndex);

          // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, ofcclFuncs[%d] returns, sharedCollCtx.saveCtx7Quit = %d, coll_id = %d", thrdCudaDev, bid, tid, sharedCollCtx.workElem.header.funcIndex, sharedCollCtx.saveCtx7Quit, collId);
          // __syncwarp();  // ！！！！！！为了打印log加的！！！

          // ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit);

          // OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, ofcclBarrier returns", thrdCudaDev, bid, tid);
        }
        *(blkStatus.barrierCnt + 0 + 13 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
        *(blkStatus.barrierCnt + 1 + 13 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = wid;
        *(blkStatus.barrierCnt + 2 + 13 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
        *(blkStatus.barrierCnt + 3 + 13 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;
        if (tid == 0) {
          // 以下的if-else事实上是和当前的工作的warp相关的，不过里边只有tid==0干活，应该也没啥影响。
          if (sharedCollCtx.saveCtx7Quit == 1) {
            saveExcutingCollCtx(thrdCudaDev, globalCollCtx4Blk7Coll, thrdLimit, collId);
          } else {
            
            // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, complete collId = %d", thrdCudaDev, bid, tid, collId);
            // // OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, complete collId = %d", thrdCudaDev, bid, tid, collId);
            // // ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit); // ！！！！！！为了打印log加的！！！！

            // atomicAdd(&sharedCollCtx.numDoneThrds, 1); // 有了线程同步，感觉这个变量在跑到底的时候没啥用。
            // 把对CQ的操作当做循环任务列表的附加动作吧，完成一个集合通信，就操作相应的CQE。
            // 完成的时候才进行下边的调用，只是保存上下文退出不应该调用。
            manipulateCQ7ResetDoneColl(thrdCudaDev, collId, cq, globalCqes, globalCollCtx4Blk7Coll);
            // 对于完成执行的集合通信应该不用把shmem里的collCtx写回到global mem里边，sendbuff/recvbuff等下次的SQE传过来，剩下的其他都是些静态配置项。
            // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> coll_id = %d done", thrdCudaDev, bid, tid, collId);
          }
          // __threadfence_block();
        }
        *(blkStatus.barrierCnt + 0 + 7 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
        // __syncthreads();
        // __syncwarp();
        ofcclBarrier(1);
        *(blkStatus.barrierCnt + 1 + 7 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
        *(blkStatus.barrierCnt + 2 + 7 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
        *(blkStatus.barrierCnt + 3 + 7 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;

        // if (sharedCollCtx.saveCtx7Quit == 0) {
        //   OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> handle collId = %d, blkStatus.numActiveColls = %d, totalVolunteerQuitCnt = %llu", thrdCudaDev, bid, threadIdx.x, collId, blkStatus.numActiveColls, blkStatus.totalVolunteerQuitCnt);
        // }
      }
    }

    *(blkStatus.barrierCnt + 0 + 16 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = i;
    *(blkStatus.barrierCnt + 1 + 16 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = collCount;
    *(blkStatus.barrierCnt + 2 + 16 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
    *(blkStatus.barrierCnt + 3 + 16 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;
    
    *(blkStatus.barrierCnt + 0 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  }
  *(blkStatus.barrierCnt + 1 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, i = %d, blkStatus.numActiveColls = %d, totalVolunteerQuitCnt = %llu", thrdCudaDev, bid, threadIdx.x, i, blkStatus.numActiveColls, blkStatus.totalVolunteerQuitCnt);
  ofcclBarrier(1); // TODO: debug
  *(blkStatus.barrierCnt + 2 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;
  *(blkStatus.barrierCnt + 3 + 11 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = currNumActiveColls;

  return turn;
}

// TODO: 考虑在按需启停的场景下，会多次启动，执行上会不会有什么变化。
__global__ void daemonKernel(SQ *sq, CQ *cq, int thrdCudaDev, int collCount, CQE *globalCqes, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx, int *globalVolunteerQuit, int *finallyQuit, BlkStatus *globalBlkStatus, unsigned long long int *barrierCnt) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  if (tid == 0) {
    blkStatus.quit = 0;
    blkStatus.barrierCnt = barrierCnt;
    BlkStatus *myGlobalBlkStatus = globalBlkStatus + bid;
    if (myGlobalBlkStatus->hasVolunteerQuitted == 0) {
      blkStatus.numActiveColls = 0;
      blkStatus.currActiveCollId = -1;
      blkStatus.sqReadFrontier = 0;
      blkStatus.hasVolunteerQuitted = 0;

      blkStatus.totalCtxSwitchCnt = 0;
      blkStatus.totalVolunteerQuitCnt = 0;
    } else { // 从volunteer quit恢复回来
      blkStatus.numActiveColls = myGlobalBlkStatus->numActiveColls;
      blkStatus.currActiveCollId = myGlobalBlkStatus->currActiveCollId;
      blkStatus.sqReadFrontier = myGlobalBlkStatus->sqReadFrontier;
      blkStatus.hasVolunteerQuitted = 1;

      blkStatus.totalCtxSwitchCnt = myGlobalBlkStatus->totalCtxSwitchCnt;
      blkStatus.totalVolunteerQuitCnt = myGlobalBlkStatus->totalVolunteerQuitCnt;
    }

    // 每次kernel启动后重置volunteer quit的控制信息
    if (bid == 0) {
      atomicExch(globalVolunteerQuit, 0);
    }
    // __threadfence_block();
  }
  // // *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1; // 这个会有内存问题，因为其他线程还没看到0号线程的更新。
  // __syncthreads();
  // __syncwarp();
  ofcclBarrier(1);
  *(blkStatus.barrierCnt + 1 + 8 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;
  
  // OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, daemonKernel starts, blkStatus.totalVolunteerQuitCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.totalVolunteerQuitCnt, blkStatus.numActiveColls);
  // __syncwarp(); // ！！！！！！为了打印log加的！！！！
  
  // int tempRound = 0;
  int turn = 0;

  turn = initContexts(thrdCudaDev, collCount, globalBlkCount4Coll, globalThrdCount4Coll, globalCollIds, globalDevComm7WorkElems, globalBlk2CollId2CollCtx, turn);
  
  int checkSQFailCnt = 0;
  while (true) {
    for (int i = 0; i < TRAVERSE_TIMES; i++) {
      // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, before traverseGlobalCollCtx, (%d / %d), blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, i, TRAVERSE_TIMES, blkStatus.numActiveColls);

      // 这个看起来是解决-M 200卡住这个问题的关键。
      // __syncwarp(); // ！！！！！！为了打印log加的！！！！

      turn = traverseGlobalCollCtx(thrdCudaDev, globalBlk2CollId2CollCtx, collCount, cq, globalCqes, turn);
      
      // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, traverseGlobalCollCtx return, (%d / %d)", thrdCudaDev, blockIdx.x, tid, i, TRAVERSE_TIMES);
      // __syncwarp(); // ！！！！！！为了打印log加的！！！！
      *(blkStatus.barrierCnt + 0 + 12 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;  
    }
    
    // OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> before checkSQ, blkStatus.numActiveColls = %d, totalVolunteerQuitCnt = %llu", thrdCudaDev, bid, threadIdx.x, blkStatus.numActiveColls, blkStatus.totalVolunteerQuitCnt);
    
    *(blkStatus.barrierCnt + 1 + 12 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;  
    *(blkStatus.barrierCnt + 2 + 12 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = TRAVERSE_TIMES;  

    if (tid == 0) {

      checkSQ(thrdCudaDev, sq, globalBlk2CollId2CollCtx, &checkSQFailCnt, finallyQuit);
      
      // 只有0号线程才会执行checkSQ，自然只有0号线程才会更改checkSQFailCnt，并且进行相应调整。

      // checkSQFailCnt = 0; // TODO: 禁止主动退出；本来想用ParseBooleanFromEnv这样的方法用env控制，不过是device函数，还是算了。

      if (checkSQFailCnt > TOLERANT_FAIL_CHECK_SQ_CNT) {
        // 主动退出。
        // 区别对待0号blk和其他。0号决定退出，其他才可以退。
        if (bid == 0) {
          atomicExch(globalVolunteerQuit, 1);
        }
        __threadfence(); // device. // TODO: 是否真的必要？应该是有必要的。
        if (*globalVolunteerQuit == 1) { // 0和其他blk都进入这里。编号较大的blk在0号block没退出的情况下，可以继续循环执行checkSQ，可以发现blkStatus.sqReadFrontier < sq->head，从而将checkSQFailCnt置零。
          BlkStatus *myGlobalBlkStatus = globalBlkStatus + bid;

          myGlobalBlkStatus->hasVolunteerQuitted = 1;
          blkStatus.quit = 1;
          ++blkStatus.totalVolunteerQuitCnt;

          // 保存blkstatus
          myGlobalBlkStatus->numActiveColls = blkStatus.numActiveColls;
          myGlobalBlkStatus->currActiveCollId = blkStatus.currActiveCollId;
          myGlobalBlkStatus->sqReadFrontier = blkStatus.sqReadFrontier;
          myGlobalBlkStatus->totalCtxSwitchCnt = blkStatus.totalCtxSwitchCnt;
          myGlobalBlkStatus->totalVolunteerQuitCnt = blkStatus.totalVolunteerQuitCnt;

          // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, Volunteer Quit, checkSQFailCnt = %d, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, checkSQFailCnt, blkStatus.numActiveColls);
        }
      }
      // __threadfence_block();
    }

    *(blkStatus.barrierCnt + 0 + 9 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;  
    // __syncthreads(); // 主要是等0号线程操作checkSQ
    // __syncwarp();
    ofcclBarrier(2);
    *(blkStatus.barrierCnt + 1 + 9 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) += 1;  
    *(blkStatus.barrierCnt + 2 + 9 * BARCNT_INNER_SIZE + tid * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;  
    
    // daemonKernel一开始这个数组用不上，可以用来记点其他信息
    *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 33 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.totalCtxSwitchCnt;
    *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 34 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.totalVolunteerQuitCnt;
    *(blkStatus.barrierCnt + 0 + 8 * BARCNT_INNER_SIZE + 35 * NUM_BARRIERS * BARCNT_INNER_SIZE + blockIdx.x * blockDim.x * NUM_BARRIERS * BARCNT_INNER_SIZE) = blkStatus.numActiveColls;

    // OFCCL_LOG_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, after checkSQ, blkStatus.sqReadFrontier = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.sqReadFrontier, blkStatus.numActiveColls);
    // OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> checkSQ returns, blkStatus.numActiveColls = %d, totalVolunteerQuitCnt = %llu", thrdCudaDev, bid, threadIdx.x, blkStatus.numActiveColls, blkStatus.totalVolunteerQuitCnt);
    // __syncwarp(); // ！！！！！！为了打印log加的！！！！
    
    if (blkStatus.quit == 1) {
      if (*finallyQuit == 1) { // 是因为每个线程都读这个global变量，所以导致了分化吗？好像不是，把这里和traverseGlobalCollCtx之前的syncthreads都注释掉，看起来又卡住了。
        OFCCL_LOG_THRD_0(OFCCL_FINAL_OR_VOLUNTEER_QUIT, "Rank<%d> Blk<%d> Thrd<%d> collCount=%d, totalCtxSwitchCnt=%llu, totalVolunteerQuitCnt=%llu", thrdCudaDev, bid, tid, collCount, blkStatus.totalCtxSwitchCnt, blkStatus.totalVolunteerQuitCnt);
        // OFCCL_LOG_BLK_0_THRD_0(OFCCL_FINAL_OR_VOLUNTEER_QUIT, "Rank<%d> Blk<%d> Thrd<%d> collCount=%d, totalCtxSwitchCnt=%llu", thrdCudaDev, bid, tid, collCount, blkStatus.totalCtxSwitchCnt);
      }
      
      // OFCCL_LOG_WARP_HEAD(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, Volunteer Quit, blkStatus.numActiveColls = %d", thrdCudaDev, blockIdx.x, tid, blkStatus.numActiveColls);

      return;
    }
  }
}