#include "hip/hip_runtime.h"
#include "enqueue_ofccl_dev.h"

// TODO: nccl最新的代码里，这部分的设计和实现都变了。
// 
// Copy src to dst and fill extra size with zeroes
// 这个是保证在一次调用复制完最多512B，并且以16B为单位。
// 这个不要求src dst同一类型
template<typename Tdst, typename Tsrc>
static __device__ void copyToShmemOneShot(Tdst *dst, Tsrc const *src, int tid, int nthreads) { // nccl的这个的函数签名里有个nthreads参数，但是并没有用，应该是为了和下边那个作区分，现在我们可以区分开了，反而带上nthreads是区分不开的。
  static_assert(sizeof(Tdst)%(2*sizeof(uint64_t)) == 0 && sizeof(Tsrc)%(2*sizeof(uint64_t)) == 0,
      "copyToShmemOneShot needs sizes which are multiple of 16B");
  static_assert(sizeof(Tdst) >= sizeof(Tsrc), "Tdst size is too small");
  static_assert(sizeof(Tdst) <= WARP_SIZE*2*sizeof(uint64_t), "copyToShmemOneShot limited to 512B to make sure it can always be done in one cycle");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  uint64_t *shmemPtr = shmemCvtPtr_ofccl(d); // 由于这个地方，这个函数只能用于dst是shmem的情况了。
  int offset = 2*tid;
  uint64_t v0, v1;
  if (offset >= sizeof(Tsrc)/sizeof(uint64_t)) {
    v0 = v1 = 0ULL;
  } else {
    v0 = s[offset] ; v1 = s[offset+1];
  }
  if (offset < sizeof(Tdst)/sizeof(uint64_t)) storeShmem128_ofccl(shmemPtr+offset, v0, v1);
}

// 这个可以直接用到任意一轮搞不完的数据结构的复制吧。
// 这个要求src dst同一类型。
// turn的作用：   
template<typename T>
static __device__ int copyToShmemLoop(T *dst, T const *src, int tid, int nthreads, int turn=0) {
  static_assert(sizeof(uint64_t) <= alignof(T), "Uhoh");
  uint64_t *d = reinterpret_cast<uint64_t*>(dst);
  uint64_t const *s = reinterpret_cast<uint64_t const*>(src);
  int t = tid - turn;
  if (t < 0) t += nthreads;
  int n = sizeof(T)/sizeof(uint64_t); // n 代表要复制的数据结构包含了几个8Byte

  int delta = (n + WARP_SIZE-1) & -WARP_SIZE; // round up to warp lane 0; 要把n和WARP_SIZE处理对齐了。
  //  32 = 0000 0000 0010 0000
  // -32 = 1111 1111 1110 0000，低位不变，高位都置1。大一的东西忘却了。。
  // 所以delta相当于n相对于32的“向上取整”，即向上取到32的整数倍。

  if (delta < nthreads) { // 总的要传的 8Byte 的个数小于blockDim.x（我们的case里是thrdLimit）
    turn += delta;
    if (turn >= nthreads) turn -= nthreads; // 在第一次调用里这个不会成立，应该是为了后续的调用使用
  }
  else
    turn = 0; // 如果总的要传的 8Byte 的个数超过了blockDim.x，那就不用管turn了。所以turn就是为了雨露均沾，让所有线程都干活

  n -= t; // 对每个线程来说，砍掉比tid小的几项，不用自己管。
  d += t; // 对每个线程来说，自己从tid的偏移量开始管。
  s += t;
  #pragma unroll // 指示要循环展开。
  for (int i=0; i < divUp(sizeof(T), WARP_SIZE*sizeof(uint64_t)); i++) {
    if (n > 0) {
      *d = *s;
      d += nthreads;
      s += nthreads;
      n -= nthreads; // “一轮”完成 nthreads个8 Byte的复制。
    }
  }
  return turn;
}

// 这个的目的应该是在“切片并行复制”之后，恢复标量的语义
static __device__ void ofcclRedopPtrDeref(struct ncclWorkElem* we) {
  if (we->header.type != ncclWorkTypeUnused && we->redOpArgIsPtr) {
    /* redOpArg is a pointer to the scalar value, so we'll dereference it
     * here so that redOpArg holds the bits of the scalar going forward.
     * The tricky thing is we don't know its type T since that's encoded in
     * the funcIndex. Because it would be difficult to get sizeof(T) from
     * funcIndex, we'll cheat and just dereference the largest possible size
     * given the alignment of the pointer. We might be reading in more bytes
     * than we need but that's harmless.
     */
    if (we->redOpArg%2 != 0)
      we->redOpArg = *reinterpret_cast<uint8_t*>(we->redOpArg);
    else if (we->redOpArg%4 != 0)
      we->redOpArg = *reinterpret_cast<uint16_t*>(we->redOpArg);
    else if (we->redOpArg%8 != 0)
      we->redOpArg = *reinterpret_cast<uint32_t*>(we->redOpArg);
    else
      we->redOpArg = *reinterpret_cast<uint64_t*>(we->redOpArg);
  }
}

// share mem用超了。
// TODO: 可以不同的algo、proto使用不同的数据类型，不过可以看看是不是有意义
__shared__ CollCtx sharedCollCtx; // 不能static，primitives要用

static __shared__ BlkStatus blkStatus;
// TODO: 下边这几个可以尝试用constant，先不急
static __shared__ int sharedCollIds[MAX_LENGTH]; // prepareColl会接受用户传进来的collId，而prepareColl工作在每个rank上，我们不能假设各个rank会收到连续的collId，所以用一个数组把收到的collId整理起来，其实相当于是维护了一个map，但是cuda上没有map，只好用这种方式
static __shared__ int sharedBlkCount4Coll[MAX_LENGTH];
static __shared__ int sharedThrdCount4Coll[MAX_LENGTH];

static __device__ int sqRead(SQ *sq, unsigned long long int sqReadFrontier, SQE *target, int thrdCudaDev) {
  int bid = blockIdx.x;
  int sqeCollId;
  
  // int tid = threadIdx.x;
  // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> enter sqRead, sqHead=%llu, sqTail=%llu, empty=%d, RingBuffer_get(sq, sqReadFrontier)->counter=%d, RingBuffer_get(sq, sqReadFrontier)->collId=%d, RingBuffer_get(sq, sqReadFrontier)->quit=%d, RingBuffer_get(sq, sqReadFrontier)->logicHead=%d, GetLogicFrontier(sq, sqReadFrontier)=%llu", thrdCudaDev, bid, tid, RingBuffer_logic_head(sq), RingBuffer_logic_tail(sq), RingBuffer_empty(sq), RingBuffer_get(sq, sqReadFrontier)->counter, RingBuffer_get(sq, sqReadFrontier)->collId, RingBuffer_get(sq, sqReadFrontier)->quit, RingBuffer_get(sq, sqReadFrontier)->logicHead, GetLogicFrontier(sq, sqReadFrontier));
  if (RingBuffer_empty(sq)) {
    return -1;
  }
  // 先读过来，然后再判断，最后更新状态：sqe->counter; 以及在恰当的时候commit read
  *target = *RingBuffer_get(sq, sqReadFrontier);
  if (target->quit) {
    // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Get quit", thrdCudaDev, bid, tid);
    return 0;
  }

  // 先判断一下相应的collId是不是该自己的bid处理，不该自己处理直接返回-1
  sqeCollId = target->collId;
  // OFCCL_LOG(OFCCL, "Blk<%d>, Thrd<%d> sharedBlkCount4Coll[%d]=%d", thrdCudaDev, bid, tid, sqeCollId, sharedBlkCount4Coll[sqeCollId]);
  if (bid >= sharedBlkCount4Coll[sqeCollId]) {
    return -1; // TODO: 所有block都应处理，blkLimit以外的直接标记完成就好。
  } else {
    // 自己读到之后，更新相应的counter；至于读到的sqe对应的collId是不是该自己处理，是caller的事。
    // 如果发现自己读完之后，所有block都读了，那么commit read
    // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> PREPARE to increase counter(curr=%d) for sqe of collId %d", thrdCudaDev, bid, tid, RingBuffer_get(sq, sqReadFrontier)->counter, sqeCollId);
    int old_counter = atomicAdd(&(RingBuffer_get(sq, sqReadFrontier)->counter), 1);
    __threadfence_system();
    // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> increase counter to %d for sqe of collId %d", thrdCudaDev, bid, tid, old_counter + 1, sqeCollId);
    
    if (old_counter + 1 == sharedBlkCount4Coll[sqeCollId]) {
      
      unsigned long long int old_head = atomicAdd(&sq->head, 1);

      __threadfence_system();
      // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sqe of collId %d commit read, new sqHead is %llu", thrdCudaDev, bid, tid, sqeCollId, old_head + 1);
    }
  }
  
  return 0;
}

static __device__ int cqWrite(CQ *cq, CQE *cqe, int thrdCudaDev) {
  // int bid = blockIdx.x;
  // int tid = threadIdx.x;
  // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> enter cqRead, RingBuffer_full(cq)=%d, cqHead=%llu, cqTail=%llu", thrdCudaDev, bid, tid, RingBuffer_full(cq), RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
  if (RingBuffer_full(cq)) {
    // not an error; caller keeps trying.
    return -1;
  }

  *RingBuffer_get_tail(cq) = *cqe;

  __threadfence_system();

  atomicAdd(&cq->tail, 1); // uint64, 一往无前++
  // RingBuffer_commit_write(cq, 1);

  return 0;
}


static __device__ int initContexts(int thrdCudaDev, int collCount, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx, int turn) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int nthreads = blockDim.x;
  // 构建任务列表
  for (int i = 0; i < collCount; i++) {
    int collId = sharedCollIds[i] = globalCollIds[i];
    // 以下这两个变量会限制很多行为。
    int blkLimit = sharedBlkCount4Coll[collId] = globalBlkCount4Coll[collId];
    sharedThrdCount4Coll[collId] = globalThrdCount4Coll[collId];

    // 每个block一份globalShmem
    CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;

    // ***** 移植ncclKernel的逻辑 *****
    if (bid < blkLimit) {
      ncclDevComm *comm = globalDevComm7WorkElems[collId].comm;
      turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->comm), comm, tid, nthreads, turn);
      // 一个奇技淫巧：get address of channel without incurring indirect load from ncclDevComm::channels
      // 这里通过bid选择了合适的channel，很多集合通信真正执行时用到的硬件信息就存在channel里边。
      ncclChannel *channel = &((ncclDevCommAndChannels*)comm)->channels[bid];
      turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->channel), channel, tid, nthreads, turn); // 尝试使用oneshot，会报错warp misaligned，所以看来必须用loop。

      // nccl中限制只在bid=0里进行这样的拷贝，对于ofccl而言，ofcclShmem就是任务列表，所以对于所有的线程，我们都把同样的work存进去；
      turn = copyToShmemLoop(&(globalCollCtx4Blk7Coll->work.elems[0]), &(globalDevComm7WorkElems[collId].first), tid, nthreads, turn); // nccl 2.12里边这地方用copyToShmemOneShot进行拷贝，但是oneShot的实现使用了与shared mem相关的内联汇编，所以这里也使用loop进行拷贝。
      // nccl中接下来要处理channel.workFifoDev，然而对于目前的ofccl，只处理first就好，channel.workFifoDev不会有其他任务了。
    }
    __syncthreads();
    if (bid < blkLimit) {
      if (globalCollCtx4Blk7Coll->work.header.type == ncclWorkTypeColl) {
        // #define NCCL_MAX_WORK_ELEMENTS (NCCL_WORK_SIZE / sizeof(struct ncclWorkElem))=512/64=8
        // 原来这个写法，应该是想修改we->redOpArg，不过修改we->redOpArg一个线程就够了，所以让理论上最多的线程来工作，咱们保留就好。
        if (tid < NCCL_MAX_WORK_ELEMENTS) ofcclRedopPtrDeref(&(globalCollCtx4Blk7Coll->work.elems[tid]));
      } // 目前不用考虑其他ncclWorkType
    }
    __syncthreads();

    if (bid < blkLimit && tid == 0) {
      globalCollCtx4Blk7Coll->executing = 0;
      // globalCollCtx4Blk7Coll->numDoneThrds = 0;
      
      globalBlk2CollId2CollCtx->saveCtx7Quit = 0;
      globalBlk2CollId2CollCtx->slice4SimpleGenericOp = 0;
      globalBlk2CollId2CollCtx->offset4SimpleGenericOp = 0;

      globalBlk2CollId2CollCtx->currentStep4RingAllReduce = 0;
      globalBlk2CollId2CollCtx->gridOffset4RingAllReduce = 0;
      // 事实上下边这3个变量不需要初始值
      globalBlk2CollId2CollCtx->offset4RingAllReduce = 0;
      globalBlk2CollId2CollCtx->nelem4RingAllReduce = 0;
      globalBlk2CollId2CollCtx->chunk4RingAllReduce = 0;
      // OFCCL_LOG(OFCCL, "nthreads: globalCollCtx4Blk7Coll->work.elems[0].nWarps*WARP_SIZE=%d, thrdLimit=%d", globalCollCtx4Blk7Coll->work.elems[0].header.nWarps*WARP_SIZE, thrdLimit);
    }
    __syncthreads();
  }
  return turn;
}

static __device__ void checkSQ(int thrdCudaDev, SQ *sq, CollCtx *globalBlk2CollId2CollCtx) {
  int bid = blockIdx.x;
  // int tempThrdCudaDev = thrdCudaDev;
  
  SQE target;
  // TODO: really need system?? 之后可以看看__threadfence()会不会提高性能。
  __threadfence_system(); // make sure read new head.

  OFCCL_LOG_BLK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, sq @ %p", thrdCudaDev, bid, threadIdx.x, sq);
  OFCCL_LOG_BLK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, sq->head = %llu", thrdCudaDev, bid, threadIdx.x, sq->head);

  if (blkStatus.sqReadFrontier < sq->head) {
    // 如果当前bid比较大，一些SQE不需要这个block处理，就会跳过。导致当前block的frontier小于head。
    // 不给sqRead增加返回值种类；否则会增加无谓的sqRead调用、增加访存次数。
    blkStatus.sqReadFrontier = sq->head;
  }

  // 能读到，假如是正常SQE，把信息在任务列表里记录一下；假如是quit，那也记录一下
  // 读不到新东西那就算了
  if (RingBuffer_logic_tail(sq) == GetLogicFrontier(sq, blkStatus.sqReadFrontier) || sqRead(sq, blkStatus.sqReadFrontier, &target, thrdCudaDev) == -1) {
    return;
  } else {
    blkStatus.sqReadFrontier++;
    if (target.quit) {
      blkStatus.quit = 1;
      // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> Main Thrd of Blk quit", thrdCudaDev, bid, threadIdx.x);
      return;
    }

    // 正常读到了SQE的话，需要往global的globalBlk2CollId2CollCtx表项里边写入，更新blkStatus.numActiveColls
    int newActiveCollId = target.collId;
    int blkLimit = sharedBlkCount4Coll[newActiveCollId];
    if (bid < blkLimit) {
      CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + newActiveCollId;
      globalCollCtx4Blk7Coll->executing = 1;
      globalCollCtx4Blk7Coll->work.elems[0].sendbuff = target.sendbuff;
      globalCollCtx4Blk7Coll->work.elems[0].recvbuff = target.recvbuff;
      
      // IF_CHECK 这个没用
      // float *sendptr = (float *)target.sendbuff;
      // for (int i = 0; i < buffPrintNum; i++) {
      //   OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sendbuff @%p sendbuff[%d]=%f", thrdCudaDev, bid, threadIdx.x, i, target.sendbuff, *(sendptr + i));
      // }
      
      // block的0号线程操作shmem，不用原子操作
      blkStatus.numActiveColls += 1;
      // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> get collId %d, blkStatus.sqReadFrontier updates to %llu, blkStatus.numActiveColls = %d", thrdCudaDev, bid, threadIdx.x, target.collId, GetLogicFrontier(sq, blkStatus.sqReadFrontier), blkStatus.numActiveColls);
      __threadfence_block();
    }
  }
}

static __device__ void manipulateCQ(int thrdCudaDev, int doneCollId, CQ *cq, CQE *globalCqes) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blkLimit = sharedBlkCount4Coll[doneCollId];
  int thrdLimit = sharedThrdCount4Coll[doneCollId];
  
  if (bid < blkLimit && tid == 0) {
    // 协调所有blk，发现所有blk都完成，最后一个blk发送CQE
    int old_counter = atomicAdd(&(globalCqes[doneCollId].counter), 1);
    __threadfence(); // cqes在global memory里边，全部thread关心。

    if (old_counter + 1 == sharedBlkCount4Coll[doneCollId]) {
      atomicExch(&globalCqes[doneCollId].counter, 0);
      while (cqWrite(cq, globalCqes + doneCollId, thrdCudaDev) == -1) {
        // tempRound++;
        // if(tempRound % tempPrintRound == 0) {
        //   OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> cqWrite fail, RingBuffer_full(cq)=%d, cqHead=%llu, cqTail=%llu", thrdCudaDev, bid, tid, RingBuffer_full(cq), RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
        // }

      }
      // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> insert CQE for collId %d, cqHead=%llu, cqTail=%llu", thrdCudaDev, bid, tid, doneCollId, RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
      // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> insert CQE for collId %d, cqHead=%llu, cqTail=%llu", thrdCudaDev, bid, tid, doneCollId, RingBuffer_logic_head(cq), RingBuffer_logic_tail(cq));
      __threadfence();
    }
  }
  ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit);
}

static __device__ int loadCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int collId, int turn) {
  // int bid = blockIdx.x;
  int tid = threadIdx.x;
  int nthreads = blockDim.x;

  turn = copyToShmemLoop(&sharedCollCtx.comm, &(globalCollCtx4Blk7Coll->comm), tid, nthreads, turn);
  turn = copyToShmemLoop(&sharedCollCtx.channel, &(globalCollCtx4Blk7Coll->channel), tid, nthreads, turn);
  // copyToShmemOneShot(&sharedCollCtx.work, &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads); // TODO: 用了这个会报错misaligned，就先loop吧
  turn = copyToShmemLoop(&sharedCollCtx.work.elems[0], &(globalCollCtx4Blk7Coll->work.elems[0]), tid, nthreads, turn);
  __syncthreads(); // 全部线程都执行，可以使用这个同步。
  
  if (sharedCollCtx.work.header.type == ncclWorkTypeColl) {
    if (tid < NCCL_MAX_WORK_ELEMENTS) ofcclRedopPtrDeref(&(sharedCollCtx.work.elems[tid]));
  } // 目前不用考虑其他ncclWorkType
  if (tid == 0) {
    // TODO: 目前只有simple ring allreduce，之后考虑通用性和扩展性。
    // 加载algo、proto、func相关的运行上下文。

    // sharedCollCtx.saveCtx7Quit = globalCollCtx4Blk7Coll->saveCtx7Quit; // 这个看起来也可以充当标记是否是跑了一半的标记位
    sharedCollCtx.saveCtx7Quit = 0; // 每次加载的时候，重置。
    sharedCollCtx.slice4SimpleGenericOp = globalCollCtx4Blk7Coll->slice4SimpleGenericOp;
    sharedCollCtx.offset4SimpleGenericOp = globalCollCtx4Blk7Coll->offset4SimpleGenericOp;

    // sharedCollCtx.totalSteps4RingAllReduce = 2 * sharedCollCtx.comm.nRanks - 1;
    sharedCollCtx.currentStep4RingAllReduce = globalCollCtx4Blk7Coll->currentStep4RingAllReduce;
    sharedCollCtx.gridOffset4RingAllReduce = globalCollCtx4Blk7Coll->gridOffset4RingAllReduce;
    sharedCollCtx.offset4RingAllReduce = globalCollCtx4Blk7Coll->offset4RingAllReduce;
    sharedCollCtx.nelem4RingAllReduce = globalCollCtx4Blk7Coll->nelem4RingAllReduce;
    sharedCollCtx.chunk4RingAllReduce = globalCollCtx4Blk7Coll->chunk4RingAllReduce;
  }
  __syncthreads();
  
  return turn;
}

static __device__ void resetDoneColl(int thrdCudaDev, int doneCollId, CollCtx *globalCollCtx4Blk7Coll) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int blkLimit = sharedBlkCount4Coll[doneCollId];
  int thrdLimit = sharedThrdCount4Coll[doneCollId];

  if (bid < blkLimit && tid == 0) {
    blkStatus.numActiveColls -= 1;
    blkStatus.currActiveCollId = -1;
    globalCollCtx4Blk7Coll->executing = 0;
    
    /* IF_CHECK 如果也好检查对错，把下边露出来 */

    // float *sendptr = (float *)sharedCollCtx.work.elems[0].sendbuff;
    // float *ptr = (float *)sharedCollCtx.work.elems[0].recvbuff;
    // for (int i = buffPrintStart; i < buffPrintStart+buffPrintNum; i++) {
    //   OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> sendbuff @ %p sendbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].sendbuff, i, *(sendptr + i));
    //   OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> recvbuff @ %p recvbuff[%d]=%f", thrdCudaDev, bid, tid, sharedCollCtx.work.elems[0].recvbuff, i, *(ptr + i));
    // }

  }
  ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit);
}

static __device__ void saveExcutingCollCtx(int thrdCudaDev, CollCtx *globalCollCtx4Blk7Coll, int thrdLimit) {
  int tid = threadIdx.x;
  if (tid == 0) {
    globalCollCtx4Blk7Coll->saveCtx7Quit = sharedCollCtx.saveCtx7Quit;
    globalCollCtx4Blk7Coll->slice4SimpleGenericOp = sharedCollCtx.slice4SimpleGenericOp;
    globalCollCtx4Blk7Coll->offset4SimpleGenericOp = sharedCollCtx.offset4SimpleGenericOp;
  
    globalCollCtx4Blk7Coll->currentStep4RingAllReduce = sharedCollCtx.currentStep4RingAllReduce;
    globalCollCtx4Blk7Coll->gridOffset4RingAllReduce = sharedCollCtx.gridOffset4RingAllReduce;
    globalCollCtx4Blk7Coll->offset4RingAllReduce = sharedCollCtx.offset4RingAllReduce;
    globalCollCtx4Blk7Coll->nelem4RingAllReduce = sharedCollCtx.nelem4RingAllReduce;
    globalCollCtx4Blk7Coll->chunk4RingAllReduce = sharedCollCtx.chunk4RingAllReduce;

    blkStatus.totalCtxSwitchCnt++;
    
    // int bid = blockIdx.x;
    // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, blkStatus.totalCtxSwitchCnt = %llu, blkStatus.numActiveColls = %d", thrdCudaDev, bid, tid, blkStatus.totalCtxSwitchCnt, blkStatus.numActiveColls);
  }
  ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit);
}

// TODO: 初步，我们让每个集合通信都跑到底。
static __device__ int traverseGlobalCollCtx(int thrdCudaDev, CollCtx *globalBlk2CollId2CollCtx, int collCount, CQ *cq, CQE *globalCqes, int turn) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;

  // int numSeenActiveColls = 0; // 想用这个和blkStatus.numActiveColls配合，减少下边的循环次数，不过涉及到线程间行为的同步，不太好搞定，先省略掉吧。

  __threadfence_block();
  if (blkStatus.numActiveColls == 0) {
    return turn;
  }
  for (int i = 0; i < collCount; i++) {
    // 下边这三个量是不变的。
    int collId = sharedCollIds[i];
    int blkLimit = sharedBlkCount4Coll[collId];
    int thrdLimit = sharedThrdCount4Coll[collId];

    if (bid < blkLimit) { // blk天然分化，保留这个条件
      // block内全部线程都执行：
      CollCtx *globalCollCtx4Blk7Coll = globalBlk2CollId2CollCtx + bid * MAX_LENGTH + collId;
      if (globalCollCtx4Blk7Coll->executing == 1) {
        if (tid == 0) {
          blkStatus.currActiveCollId = collId; // 0号线程修改shmem，应该不用原子操作。
          __threadfence_block();
        }

        // ***** 先准备好sharedCollCtx *****
        turn = loadCollCtx(thrdCudaDev, globalCollCtx4Blk7Coll, collId, turn); // 只load一个到shmem
        
        // 只有真正的工作线程才执行
        if (tid < thrdLimit) {
          // ***** 然后调用ofcclFunc *****
          ofcclFuncs[sharedCollCtx.work.header.funcIndex](); // 这里边的调用里不涉及__syncthreads().
          // 根据sharedCollCtx.saveCtx7Quit的情况进行不同处理。
          OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, ofcclFuncs[%d]() return", sharedCollCtx.comm.rank, blockIdx.x, threadIdx.x, sharedCollCtx.work.header.funcIndex);
  
          // if (tid == 0) {
          //   OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, ofcclFuncs returns, sharedCollCtx.saveCtx7Quit = %d", thrdCudaDev, bid, tid, sharedCollCtx.saveCtx7Quit);
          // }
          ofcclBarrier(OFCCL_SYNC_COLL_WORKER_BAR_ID, thrdLimit);

          if (sharedCollCtx.saveCtx7Quit == 1) {
            saveExcutingCollCtx(thrdCudaDev, globalCollCtx4Blk7Coll, thrdLimit);
          } else {
            // atomicAdd(&sharedCollCtx.numDoneThrds, 1); // 有了线程同步，感觉这个变量在跑到底的时候没啥用。
            // 把对CQ的操作当做循环任务列表的附加动作吧，完成一个集合通信，就操作相应的CQE。
            // 完成的时候才进行下边的调用，只是保存上下文退出不应该调用。
            manipulateCQ(thrdCudaDev, collId, cq, globalCqes);
            resetDoneColl(thrdCudaDev, collId, globalCollCtx4Blk7Coll);
            // 对于完成执行的集合通信应该不用把shmem里的collCtx写回到global mem里边，sendbuff/recvbuff等下次的SQE传过来，剩下的其他都是些静态配置项。
            // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> collId %d done", thrdCudaDev, bid, tid, collId);
          }
        }
  
        __syncthreads(); // thrdLimit内外的所有线程，都到这里同步。

      }
    }
  }

  return turn;
}

// TODO: 考虑在按需启停的场景下，会多次启动，执行上会不会有什么变化。
__global__ void daemonKernel(SQ *sq, CQ *cq, int thrdCudaDev, int collCount, CQE *globalCqes, int *globalBlkCount4Coll, int *globalThrdCount4Coll, int *globalCollIds, DevComm7WorkElem *globalDevComm7WorkElems, CollCtx *globalBlk2CollId2CollCtx) {
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  SQ *localSq = sq;
  
  // int tempRound = 0;
  int turn = 0;

  turn = initContexts(thrdCudaDev, collCount, globalBlkCount4Coll, globalThrdCount4Coll, globalCollIds, globalDevComm7WorkElems, globalBlk2CollId2CollCtx, turn);
  
  if (tid == 0) {
    blkStatus.quit = 0;
    blkStatus.numActiveColls = 0;
    blkStatus.currActiveCollId = -1;
    blkStatus.sqReadFrontier = 0;
    blkStatus.totalCtxSwitchCnt = 0;
    // __threadfence_block();
  }
  __syncthreads();
  while (true) {
    for (int i = 0; i < TRAVERSE_TIMES; i++) {
      turn = traverseGlobalCollCtx(thrdCudaDev, globalBlk2CollId2CollCtx, collCount, cq, globalCqes, turn);
      OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, traverseGlobalCollCtx return, (%d / %d)", sharedCollCtx.comm.rank, blockIdx.x, tid, i, TRAVERSE_TIMES);
    }
    if (tid == 0) {
      // OFCCL_LOG(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, before checkSQ, sq @ %p, blkStatus.numActiveColls=%d, blkStatus.currActiveCollId=%d, blkStatus.totalCtxSwitchCnt=%d", thrdCudaDev, bid, tid, localSq, blkStatus.numActiveColls, blkStatus.currActiveCollId, blkStatus.totalCtxSwitchCnt);
      
      OFCCL_LOG_BLK_0_THRD_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d>, before checkSQ, sq @ %p", sharedCollCtx.comm.rank, blockIdx.x, tid, localSq);
      checkSQ(thrdCudaDev, localSq, globalBlk2CollId2CollCtx);
    }

    __syncthreads();
    if (blkStatus.quit == 1) {
      // OFCCL_LOG_RANK_0(OFCCL, "Rank<%d> Blk<%d> Thrd<%d> quit", thrdCudaDev, bid, tid);

      if (tid == 0) {
        OFCCL_LOG(OFCCL_FINAL, "\nRank<%d> Blk<%d> Thrd<%d> collCount=%d, totalCtxSwitchCnt=%llu", thrdCudaDev, bid, tid, collCount, blkStatus.totalCtxSwitchCnt);
      }
      return;
    }
  }
}