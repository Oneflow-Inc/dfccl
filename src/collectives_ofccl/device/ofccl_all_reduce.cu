#include "hip/hip_runtime.h"
#include "ofccl_all_reduce.h"
// #include "common.h"
#include "collectives_ofccl.h"

__global__ void try_make_kern() {
  printf("\ngridDim.x=%d, blockDim.x=%d, blockIdx=%d, threadIdx=%d\n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x);
}